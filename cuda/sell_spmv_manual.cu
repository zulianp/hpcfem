
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h> // cudaMalloc, cudaMemcpy, etc.
#include <hipsparse.h>         // cusparseSpMV
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE
#include <assert.h>

inline void cuda_check(hipError_t code, const char* file, int line, bool abort = true) {
    if (code != hipSuccess) {
        fprintf(stderr, "cuda_check: %s %s:%d\n", hipGetErrorString(code), file, line);
        assert(!code);
        if (abort) exit(code);
    }
}

#define CUDA_CHECK(ans) \
    { cuda_check((ans), __FILE__, __LINE__); }

#ifndef NDEBUG
#define DEBUG_SYNCHRONIZE()                \
    do {                                        \
        hipDeviceSynchronize();                \
        CUDA_CHECK(hipPeekAtLastError()); \
    } while (0)
#else
#define DEBUG_SYNCHRONIZE()
#endif

#define CHECK_CUDA(func)                                                       \
{                                                                              \
    hipError_t status = (func);                                               \
    if (status != hipSuccess) {                                               \
        printf("CUDA API failed at line %d with error: %s (%d)\n",             \
               __LINE__, hipGetErrorString(status), status);                  \
        return EXIT_FAILURE;                                                   \
    }                                                                          \
}

#define CHECK_CUSPARSE(func)                                                   \
{                                                                              \
    hipsparseStatus_t status = (func);                                          \
    if (status != HIPSPARSE_STATUS_SUCCESS) {                                   \
        printf("CUSPARSE API failed at line %d with error: %s (%d)\n",         \
               __LINE__, hipsparseGetErrorString(status), status);              \
        return EXIT_FAILURE;                                                   \
    }                                                                          \
}

int* load_int32_array(const char *filename, int64_t *elements_read) {
    FILE *file;
    long file_size;
    int *buffer, *device_buffer;
    size_t num_elements;

    // Open the file for reading in binary mode
    file = fopen(filename, "rb");
    if (file == NULL) {
        perror("Error opening file");
        return NULL;
    }

    // Seek to the end of the file to get its size
    fseek(file, 0, SEEK_END);
    file_size = ftell(file);
    rewind(file);

    // Calculate the number of elements based on the file size
    num_elements = file_size / sizeof(int);

    // Allocate managed memory for the buffer
    buffer = (int *) malloc(file_size);
    hipError_t err = hipMalloc((void**)&device_buffer, file_size);
    DEBUG_SYNCHRONIZE();

    if (err != hipSuccess) {
        fprintf(stderr, "Error allocating managed memory: %s\n", hipGetErrorString(err));
        fclose(file);
        return NULL;
    }

    // Read the entire content of the file into the buffer
    *elements_read = fread(buffer, sizeof(int), num_elements, file);
    if (*elements_read != num_elements) {
        perror("Error reading file");
        free(buffer);
        fclose(file);
        return NULL;
    }

    // Close the file
    fclose(file);
    hipMemcpy(device_buffer, buffer, file_size, hipMemcpyHostToDevice);
    DEBUG_SYNCHRONIZE();

    // Report the number of elements read
    printf("Number of elements read from %s: %d\n", filename, *elements_read);

    return device_buffer;
}

double* load_float64_array(const char *filename, int64_t *elements_read) {
    FILE *file;
    long file_size;
    double *buffer, *device_buffer;
    size_t num_elements;

    // Open the file for reading in binary mode
    file = fopen(filename, "rb");
    if (file == NULL) {
        perror("Error opening file");
        return NULL;
    }

    // Seek to the end of the file to get its size
    fseek(file, 0, SEEK_END);
    file_size = ftell(file);
    rewind(file);

    // Calculate the number of elements based on the file size
    num_elements = file_size / sizeof(double);

    // Allocate managed memory for the buffer
    buffer = (double *) malloc(file_size);
    hipError_t err = hipMalloc((void**)&device_buffer, file_size);
    DEBUG_SYNCHRONIZE();

    if (err != hipSuccess) {
        fprintf(stderr, "Error allocating managed memory: %s\n", hipGetErrorString(err));
        fclose(file);
        return NULL;
    }

    // Read the entire content of the file into the buffer
    *elements_read = fread(buffer, sizeof(double), num_elements, file);
    if (*elements_read != num_elements) {
        perror("Error reading file");
        free(buffer);
        fclose(file);
        return NULL;
    }

    // Close the file
    fclose(file);
    hipMemcpy(device_buffer, buffer, file_size, hipMemcpyHostToDevice);
    DEBUG_SYNCHRONIZE();

    // Report the number of elements read
    printf("Number of elements read from %s: %d\n", filename, *elements_read);

    return device_buffer;
}

int* load_int32_array_to_host(const char *filename, int64_t *elements_read) {
    FILE *file;
    long file_size;
    int *buffer;
    size_t num_elements;

    // Open the file for reading in binary mode
    file = fopen(filename, "rb");
    if (file == NULL) {
        perror("Error opening file");
        return NULL;
    }

    // Seek to the end of the file to get its size
    fseek(file, 0, SEEK_END);
    file_size = ftell(file);
    rewind(file);

    // Calculate the number of elements based on the file size
    num_elements = file_size / sizeof(int);

    // Allocate managed memory for the buffer
    buffer = (int *) malloc(file_size);

    // Read the entire content of the file into the buffer
    *elements_read = fread(buffer, sizeof(int), num_elements, file);
    if (*elements_read != num_elements) {
        perror("Error reading file");
        fclose(file);
        return NULL;
    }

    // Close the file
    fclose(file);

    // Report the number of elements read
    printf("Number of elements read from %s: %d\n", filename, *elements_read);

    return buffer;
}


int main(void) {
    // Host problem definition
    int A_num_rows      = 1;
    int A_num_cols      = 1;
    int A_nnz           = 1;
    float     alpha           = 1.0f;
    float     beta            = 0.0f;
    //--------------------------------------------------------------------------
    // Device memory management
    // int   *dA_csrOffsets, *dA_columns;
    double *dA_values, *dX, *dY;

    int64_t sellValuesSize = 0;
    int64_t elements_read;

    int *sellSliceOffsets = load_int32_array("sell_slice_offsets.i32", &elements_read);
    // CHECK_CUDA( cudaMemPrefetchAsync(sellSliceOffsets, elements_read * sizeof(int32_t), 0) )

    double *sellValues = load_float64_array("sell_values.f64", &sellValuesSize);
    // CHECK_CUDA( cudaMemPrefetchAsync(sellValues, sellValuesSize * sizeof(double), 0) )

    int *sellColInd = load_int32_array("sell_column_indices.i32", &elements_read);
    // CHECK_CUDA( cudaMemPrefetchAsync(sellColInd, elements_read * sizeof(int32_t), 0) )

    int *sellMetaInfo = load_int32_array_to_host("sell_meta.i32", &elements_read);

    int sliceSize = 2;

    A_num_rows = sellMetaInfo[0];
    A_num_cols = sellMetaInfo[0];
    A_nnz = sellMetaInfo[1];
    sliceSize = sellMetaInfo[2];

    // Timing variables
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    //--------------------------------------------------------------------------
    // CUSPARSE APIs
    hipsparseHandle_t     handle = NULL;
    hipsparseSpMatDescr_t matA;
    hipsparseDnVecDescr_t vecX, vecY;
    void*                dBuffer    = NULL;
    size_t               bufferSize = 0;
    CHECK_CUSPARSE( hipsparseCreate(&handle) )
    // Create sparse matrix A in SELL format

    CHECK_CUSPARSE( cusparseCreateSlicedEll(&matA, A_num_rows, A_num_cols, A_nnz,
                            sellValuesSize, sliceSize, sellSliceOffsets, sellColInd, sellValues,
                            HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                            HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F) )

    CHECK_CUDA( hipMalloc((void**) &dX, A_num_cols * sizeof(double)) )
    CHECK_CUDA( hipMalloc((void**) &dY, A_num_rows * sizeof(double)) )

    // Create dense vector X
    CHECK_CUSPARSE( hipsparseCreateDnVec(&vecX, A_num_cols, dX, HIP_R_64F) )
    // Create dense vector y
    CHECK_CUSPARSE( hipsparseCreateDnVec(&vecY, A_num_rows, dY, HIP_R_64F) )
    // allocate an external buffer if needed
    CHECK_CUSPARSE( hipsparseSpMV_bufferSize(
                                 handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, matA, vecX, &beta, vecY, HIP_R_64F,
                                 HIPSPARSE_SPMV_ALG_DEFAULT, &bufferSize) )
    CHECK_CUDA( hipMalloc(&dBuffer, bufferSize) )

    hipEventRecord(start);

    // execute SpMV
    CHECK_CUSPARSE( hipsparseSpMV(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, matA, vecX, &beta, vecY, HIP_R_64F,
                                 HIPSPARSE_SPMV_ALG_DEFAULT, dBuffer) )

    hipEventRecord(stop);

    // Wait for the event to complete
    hipEventSynchronize(stop);

    // Calculate elapsed time
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Time for matrix-vector multiplication: %f milliseconds\n", milliseconds);
    printf("Throughput: %f MDOF/s\n", (A_num_rows / 1e6) / (milliseconds / 1000.0));

    // destroy matrix/vector descriptors
    CHECK_CUSPARSE( hipsparseDestroySpMat(matA) )
    CHECK_CUSPARSE( hipsparseDestroyDnVec(vecX) )
    CHECK_CUSPARSE( hipsparseDestroyDnVec(vecY) )
    CHECK_CUSPARSE( hipsparseDestroy(handle) )

    //--------------------------------------------------------------------------
    // device memory deallocation
    // CHECK_CUDA( cudaFree(dBuffer) )
    // CHECK_CUDA( cudaFree(dA_csrOffsets) )
    // CHECK_CUDA( cudaFree(dA_columns) )
    // CHECK_CUDA( cudaFree(dA_values) )
    CHECK_CUDA( hipFree(dX) )
    CHECK_CUDA( hipFree(dY) )
    return EXIT_SUCCESS;
}

