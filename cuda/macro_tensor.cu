#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include <math.h>
#include <unistd.h>
#include <assert.h>

#include <hip/hip_runtime.h>
#include <mma.h>
#include <hip/hip_runtime.h>
#include <hip/hip_cooperative_groups.h>
#include <hipblas.h>
#include <cuda_pipeline.h>

// nvcc macro.cu --std=c++11 -o cargo -arch=sm_75 -g -G -lcublas
using namespace nvcuda;
using namespace cooperative_groups;

#define BLOCK_SIZE 160
typedef double real_t;

#define checkCUBLASError(call)                                                \
{                                                                           \
    hipblasStatus_t err = call;                                                 \
    if (err != HIPBLAS_STATUS_SUCCESS) { \
        printf("Error %s at %s:%d\n", cublasGetStatusString(err), __FILE__, __LINE__);\
        exit(EXIT_FAILURE);                                                 \
    }                                                                       \
}

#define checkCudaError(call)                                                \
{                                                                           \
    hipError_t err = call;                                                 \
    if (err != hipSuccess)                                                 \
    {                                                                       \
        fprintf(stderr, "CUDA Error: %s (code: %d), at %s:%d\n",            \
                hipGetErrorString(err), err, __FILE__, __LINE__);          \
        exit(EXIT_FAILURE);                                                 \
    }                                                                       \
}

#define ifLastErrorExists(msg)                                         \
{                                                                      \
    hipError_t err = hipGetLastError();                              \
    if (err != hipSuccess)                                            \
    {                                                                  \
        fprintf(stderr, "CUDA Error: %s, at %s:%d - %s\n",             \
                msg, __FILE__, __LINE__, hipGetErrorString(err));     \
        exit(EXIT_FAILURE);                                            \
    }                                                                  \
}

__device__ void print_matrix(real_t *matrix, int rows, int cols)
{
    for (int i = 0; i < rows; i++)
    {
        for (int j = 0; j < cols; j++)
        {
            printf("%f ", matrix[i * cols + j]);
        }
        printf("\n");
    }
    printf("\n");
}

int compute_nodes_number(int tetra_level)
{
    // 1 layer = 4
    // 2 layer = 10
    // 3 layer = 20
    // 4 layer = 35
    return (tetra_level + 3) * (tetra_level + 2) * (tetra_level + 1) / 6;
}

int compute_tets_number(int tetra_level)
{
    return (int) pow(tetra_level, 3);
}

__device__ real_t determinant_3x3(real_t *m) {
    // computes the inverse of a matrix m
    real_t det = m[0*3+0] * (m[1*3+1] * m[2*3+2] - m[2*3+1] * m[1*3+2]) -
        m[0*3+1] * (m[1*3+0] * m[2*3+2] - m[1*3+2] * m[2*3+0]) +
        m[0*3+2] * (m[1*3+0] * m[2*3+1] - m[1*3+1] * m[2*3+0]);
    // print_matrix(m, 3, 3);
    // printf("det(m) = %lf\n", det);
    return det;
}

__device__ __host__ int compute_tets_number_of_category(int tetra_level, int category)
{
    if (category == 0) return (tetra_level + 2) * (tetra_level + 1) * tetra_level / 6;
    if (category == 4) return (tetra_level - 2) * (tetra_level - 1) * tetra_level / 6;
    return (tetra_level + 1) * (tetra_level - 1) * tetra_level / 6;
}

__device__ void inverse_3x3_T(real_t *m, real_t *m_inv)
{
    real_t det_inv = 1.0 / determinant_3x3(m);

    m_inv[0*3+0] = (m[1*3+1] * m[2*3+2] - m[2*3+1] * m[1*3+2]) * det_inv;
    m_inv[1*3+0] = (m[0*3+2] * m[2*3+1] - m[0*3+1] * m[2*3+2]) * det_inv;
    m_inv[2*3+0] = (m[0*3+1] * m[1*3+2] - m[0*3+2] * m[1*3+1]) * det_inv;
    m_inv[0*3+1] = (m[1*3+2] * m[2*3+0] - m[1*3+0] * m[2*3+2]) * det_inv;
    m_inv[1*3+1] = (m[0*3+0] * m[2*3+2] - m[0*3+2] * m[2*3+0]) * det_inv;
    m_inv[2*3+1] = (m[1*3+0] * m[0*3+2] - m[0*3+0] * m[1*3+2]) * det_inv;
    m_inv[0*3+2] = (m[1*3+0] * m[2*3+1] - m[2*3+0] * m[1*3+1]) * det_inv;
    m_inv[1*3+2] = (m[2*3+0] * m[0*3+1] - m[0*3+0] * m[2*3+1]) * det_inv;
    m_inv[2*3+2] = (m[0*3+0] * m[1*3+1] - m[1*3+0] * m[0*3+1]) * det_inv;
}

__device__ void jacobian_to_laplacian(real_t *macro_J, real_t *micro_L, int tetra_level, int category) {
    real_t J_inv_trans[9];
    real_t micro_J[9];
    const real_t grad_ref_phi[4][3] = {
        {-1, -1, -1},
        {1, 0, 0},
        {0, 1, 0},
        {0, 0, 1}
    };
    real_t grad_phi[4][3];

    // have to match the row/col order of compute_A
    real_t u[3] = {macro_J[0], macro_J[1], macro_J[2]};
    real_t v[3] = {macro_J[3], macro_J[4], macro_J[5]};
    real_t w[3] = {macro_J[6], macro_J[7], macro_J[8]};

    if (category == 0) {
        // [u | v | w]
        for (int i = 0; i < 3; i++) {
            for (int j = 0; j < 3; j++) {
                micro_J[i * 3 + j] = macro_J[i * 3 + j] / tetra_level;
            }
        }
        // assert(determinant_3x3(micro_J) > 0);
    } else if (category == 1) {
        // [-u + w | w | -u + v + w]
        for (int i = 0; i < 3; i++) {
            micro_J[i * 3 + 0] = (-u[i] + w[i]) / tetra_level;
            micro_J[i * 3 + 1] = (w[i]) / tetra_level;
            micro_J[i * 3 + 2] = (-u[i] + v[i] + w[i]) / tetra_level;
        }
        // assert(determinant_3x3(micro_J) > 0);
    } else if (category == 2) {
        // [v | -u + v + w | w]
        for (int i = 0; i < 3; i++) {
            micro_J[i * 3 + 0] = v[i] / tetra_level;
            micro_J[i * 3 + 1] = (-u[i] + v[i] + w[i]) / tetra_level;
            micro_J[i * 3 + 2] = (w[i]) / tetra_level;
        }
        // assert(determinant_3x3(micro_J) > 0);
    } else if (category == 3) {
        // [-u + v | -u + w | -u + v + w]
        for (int i = 0; i < 3; i++) {
            micro_J[i * 3 + 0] = (-u[i] + v[i]) / tetra_level;
            micro_J[i * 3 + 1] = (-u[i] + w[i]) / tetra_level;
            micro_J[i * 3 + 2] = (-u[i] + v[i] + w[i]) / tetra_level;
        }
        // assert(determinant_3x3(micro_J) > 0);
    } else if (category == 4) {
        // [-v + w | w | -u + w]
        for (int i = 0; i < 3; i++) {
            micro_J[i * 3 + 0] = (-v[i] + w[i]) / tetra_level;
            micro_J[i * 3 + 1] = (w[i]) / tetra_level;
            micro_J[i * 3 + 2] = (-u[i] + w[i]) / tetra_level;
        }
        // assert(determinant_3x3(micro_J) > 0);
    } else if (category == 5) {
        // [-u + v | -u + v + w | v]
        for (int i = 0; i < 3; i++) {
            micro_J[i * 3 + 0] = (-u[i] + v[i]) / tetra_level;
            micro_J[i * 3 + 1] = (-u[i] + v[i] + w[i]) / tetra_level;
            micro_J[i * 3 + 2] = (v[i]) / tetra_level;
        }
        // assert(determinant_3x3(micro_J) > 0);
    }

    inverse_3x3_T(micro_J, J_inv_trans);

    for (int i = 0; i < 4; i++) {
        for (int j = 0; j < 3; j++) {
            grad_phi[i][j] = 0;
            for (int k = 0; k < 3; k++) {
                grad_phi[i][j] += J_inv_trans[j * 3 + k] * grad_ref_phi[i][k];
            }
        }
    }

    for (int i = 0; i < 4; i++) {
        for (int j = 0; j < 4; j++) {
            real_t dot_product = 0;
            for (int k = 0; k < 3; k++) {
                dot_product += grad_phi[i][k] * grad_phi[j][k];
            }
            micro_L[i * 4 + j] = dot_product * determinant_3x3(micro_J) / 6.0;
        }
    }

}

// template <typename real_t>
__global__ void cu_macro_tet4_laplacian_apply_kernel(
        const size_t n_macro_tets,
        const size_t n_micro_tets,
        const size_t n_micro_nodes,
        const size_t stride,  // Stride here represents the number of macro-elements
        int tetra_level, 
        const real_t *const macro_jacobians,
        real_t *microL,
        real_t *gathered,
        real_t *to_scatter,
        const real_t *const vecX,
        real_t *const vecY) {

    const int MATRIX_B_SIZE = 4 * 8;
    const int MATRIX_A_SIZE = 8 * 4;
    const int MATRIX_C_SIZE = 8 * 8;
    const int MATRIX_C_SHRINKED = 8 * 8;

    extern __shared__ real_t buffer[];
    // These belong to shared memory
    real_t *localX = (real_t *)buffer;
    real_t *localY = (real_t *)&localX[BLOCK_SIZE * n_micro_nodes]; 
    // real_t *microL = (real_t *)&buffer[BLOCK_SIZE * n_micro_nodes * 2]; 
    // real_t *gathered = (real_t *)&buffer[BLOCK_SIZE * (n_micro_nodes * 2 + MATRIX_A_SIZE * 6)]; 
    // real_t *gathered = (real_t *)&buffer[BLOCK_SIZE * n_micro_nodes * 2]; 

    int level = tetra_level + 1;

    real_t macro_J[9];

    // Declare the fragments
    wmma::fragment<wmma::matrix_a, 8, 8, 4, double, wmma::row_major> a_frag;
    wmma::fragment<wmma::matrix_b, 8, 8, 4, double, wmma::row_major> b_frag;
    wmma::fragment<wmma::accumulator, 8, 8, 4, double> c_frag;

    // Initialize the output to zero
    wmma::fill_fragment(c_frag, 0.0);

    for (size_t macro_tet_idx = blockIdx.x * blockDim.x + threadIdx.x; macro_tet_idx < n_macro_tets;
         macro_tet_idx += blockDim.x * gridDim.x) {

        for (int micro_node_idx = 0; micro_node_idx < n_micro_nodes; micro_node_idx++) {
            __pipeline_memcpy_async(&localX[micro_node_idx * BLOCK_SIZE + threadIdx.x],
                &vecX[micro_node_idx * stride + macro_tet_idx], sizeof(real_t));
            __pipeline_memcpy_async(&localY[micro_node_idx * BLOCK_SIZE + threadIdx.x],
                &vecY[micro_node_idx * stride + macro_tet_idx], sizeof(real_t));
        }
        __pipeline_commit();

        for (int d = 0; d < 9; d++) {
            macro_J[d] = macro_jacobians[d * stride + macro_tet_idx];
        }

        jacobian_to_laplacian(macro_J, &microL[0 * MATRIX_A_SIZE + 6 * MATRIX_A_SIZE * macro_tet_idx], tetra_level, 0);
        jacobian_to_laplacian(macro_J, &microL[1 * MATRIX_A_SIZE + 6 * MATRIX_A_SIZE * macro_tet_idx], tetra_level, 1);
        jacobian_to_laplacian(macro_J, &microL[2 * MATRIX_A_SIZE + 6 * MATRIX_A_SIZE * macro_tet_idx], tetra_level, 2);
        jacobian_to_laplacian(macro_J, &microL[3 * MATRIX_A_SIZE + 6 * MATRIX_A_SIZE * macro_tet_idx], tetra_level, 3);
        jacobian_to_laplacian(macro_J, &microL[4 * MATRIX_A_SIZE + 6 * MATRIX_A_SIZE * macro_tet_idx], tetra_level, 4);
        jacobian_to_laplacian(macro_J, &microL[5 * MATRIX_A_SIZE + 6 * MATRIX_A_SIZE * macro_tet_idx], tetra_level, 5);

        // sizeof(half) == 2
        // sizeof(float) == 4
        // sizeof(double) == 8

        // TODO: can also pass the following as params
        // NOTE: tet_1 to tet_4 have the same tet number
        int num_corner_matrices = (compute_tets_number_of_category(tetra_level, 0) + 7) / 8;
        int num_inverted_matrices = (compute_tets_number_of_category(tetra_level, 4) + 7) / 8;
        int num_pyramid_matrices = (compute_tets_number_of_category(tetra_level, 1) + 7) / 8;

        int num_total_matrices = num_corner_matrices + num_pyramid_matrices * 4 + num_inverted_matrices;
        // real_t *to_scatter = (real_t *)&buffer[BLOCK_SIZE * (n_micro_nodes * 2 + MATRIX_A_SIZE * 6 + num_total_matrices * 32)]; 

        __pipeline_wait_prior(0);

        // if (macro_tet_idx == 0) {
        //     printf("\nLaplacian of Category %d\n", 0);
        //     print_matrix(&microL[0 * MATRIX_A_SIZE + 6 * MATRIX_A_SIZE * macro_tet_idx], 8, 4);
        // }

        int vals_iter = 0;
        int base_iter = 0;

        int p = 0;
        for (int i = 0; i < level - 1; i++)
        {
            int layer_items = (level - i + 1) * (level - i) / 2;
            for (int j = 0; j < level - i - 1; j++)
            {
                for (int k = 0; k < level - i - j - 1; k++)
                {
                    int e[4] = {p, p + layer_items - j, p + level - i - j, p + 1};

                    gathered[num_total_matrices * MATRIX_B_SIZE * macro_tet_idx + base_iter + vals_iter + 0] = localX[e[0] * BLOCK_SIZE + threadIdx.x];
                    gathered[num_total_matrices * MATRIX_B_SIZE * macro_tet_idx + base_iter + vals_iter + 8] = localX[e[1] * BLOCK_SIZE + threadIdx.x];
                    gathered[num_total_matrices * MATRIX_B_SIZE * macro_tet_idx + base_iter + vals_iter + 16] = localX[e[2] * BLOCK_SIZE + threadIdx.x];
                    gathered[num_total_matrices * MATRIX_B_SIZE * macro_tet_idx + base_iter + vals_iter + 24] = localX[e[3] * BLOCK_SIZE + threadIdx.x];

                    // if (macro_tet_idx == 0) {
                    //     printf("First: [%lu] [%lu] [%lu] [%lu]\n", 
                    //         num_total_matrices * MATRIX_B_SIZE * macro_tet_idx + base_iter + vals_iter + 0, 
                    //         (num_total_matrices * MATRIX_B_SIZE * macro_tet_idx) + base_iter + vals_iter + 8,
                    //         num_total_matrices * MATRIX_B_SIZE * macro_tet_idx + base_iter + vals_iter + 16,
                    //         (num_total_matrices * MATRIX_B_SIZE * macro_tet_idx) + base_iter + vals_iter + 24);
                    //     printf("First: %lf %lf %lf %lf\n", localX[e[0] * BLOCK_SIZE + threadIdx.x], localX[e[1] * BLOCK_SIZE + threadIdx.x], 
                    //         localX[e[2] * BLOCK_SIZE + threadIdx.x], localX[e[3] * BLOCK_SIZE + threadIdx.x]);
                    //     // printf("First: %d %d %d %d\n", e[0], e[1], e[2], e[3]);
                    //     // printf("vals_iter: %d, base_iter: %d\n", vals_iter, base_iter);
                    // }

                    vals_iter += 1;
                    base_iter += (vals_iter / 8) * MATRIX_B_SIZE;
                    vals_iter = vals_iter % 8;



                    p++;
                }
                p++;
            }
            p++;
        }

        if (vals_iter > 0) base_iter += MATRIX_B_SIZE;
        vals_iter = 0;

        if (macro_tet_idx == 0) {
            printf("tet: %d, base_iter A0 %d\n", num_corner_matrices, base_iter);
            // for (int assembled_mat_idx = 0; assembled_mat_idx < base_iter / MATRIX_B_SIZE; assembled_mat_idx += 1) {
            //     printf("\nSystem 0 Matrix %d\n", assembled_mat_idx);
            //     print_matrix(&gathered[num_total_matrices * MATRIX_B_SIZE * macro_tet_idx + assembled_mat_idx * 32], 4, 8);
            // }
        }

        // vals_iter = MATRIX_B_SIZE * ((vals_iter + MATRIX_B_SIZE) / MATRIX_B_SIZE);

        // Second case

        p = 0;
        for (int i = 0; i < level - 1; i++)
        {
            int layer_items = (level - i) * (level - i - 1) / 2;
            for (int j = 0; j < level - i - 1; j++)
            {
                p++;
                for (int k = 1; k < level - i - j - 1; k++)
                {
                    int e[4] = {
                        p, 
                        p + layer_items + level - i - j - 1 + level - i - j - 1,
                        p + layer_items + level - i - j, 
                        p + layer_items + level - i - j - 1
                    };

                    gathered[num_total_matrices * MATRIX_B_SIZE * macro_tet_idx + base_iter + vals_iter + 0] = localX[e[0] * BLOCK_SIZE + threadIdx.x];
                    gathered[num_total_matrices * MATRIX_B_SIZE * macro_tet_idx + base_iter + vals_iter + 8] = localX[e[1] * BLOCK_SIZE + threadIdx.x];
                    gathered[num_total_matrices * MATRIX_B_SIZE * macro_tet_idx + base_iter + vals_iter + 16] = localX[e[2] * BLOCK_SIZE + threadIdx.x];
                    gathered[num_total_matrices * MATRIX_B_SIZE * macro_tet_idx + base_iter + vals_iter + 24] = localX[e[3] * BLOCK_SIZE + threadIdx.x];

                    vals_iter += 1;
                    base_iter += (vals_iter / 8) * MATRIX_B_SIZE;
                    vals_iter = vals_iter % 8;

                    p++;
                }
                p++;
            }
            p++;
        }

        if (vals_iter > 0) base_iter += MATRIX_B_SIZE;
        vals_iter = 0;
        // vals_iter = MATRIX_B_SIZE * ((vals_iter + MATRIX_B_SIZE) / MATRIX_B_SIZE);

        if (macro_tet_idx == 0) {
            printf("tet: %d, base_iter A1 %d\n", num_pyramid_matrices, base_iter);
        }

        // Third case
        p = 0;

        for (int i = 0; i < level - 1; i++)
        {
            int layer_items = (level - i) * (level - i - 1) / 2;
            for (int j = 0; j < level - i - 1; j++)
            {
                p++;
                for (int k = 1; k < level - i - j - 1; k++)
                {
                    int e[4] = {
                        p, 
                        p + layer_items + level - i - j,
                        p + layer_items + level - i - j - 1 + level - i - j - 1,
                        p + level - i - j
                    };

                    gathered[num_total_matrices * MATRIX_B_SIZE * macro_tet_idx + base_iter + vals_iter + 0] = localX[e[0] * BLOCK_SIZE + threadIdx.x];
                    gathered[num_total_matrices * MATRIX_B_SIZE * macro_tet_idx + base_iter + vals_iter + 8] = localX[e[1] * BLOCK_SIZE + threadIdx.x];
                    gathered[num_total_matrices * MATRIX_B_SIZE * macro_tet_idx + base_iter + vals_iter + 16] = localX[e[2] * BLOCK_SIZE + threadIdx.x];
                    gathered[num_total_matrices * MATRIX_B_SIZE * macro_tet_idx + base_iter + vals_iter + 24] = localX[e[3] * BLOCK_SIZE + threadIdx.x];

                    vals_iter += 1;
                    base_iter += (vals_iter / 8) * MATRIX_B_SIZE;
                    vals_iter = vals_iter % 8;

                    p++;
                }
                p++;
            }
            p++;
        }

        if (vals_iter > 0) base_iter += MATRIX_B_SIZE;
        vals_iter = 0;

        // if (macro_tet_idx == 0) {
        //     printf("tet: %d, base_iter A2 %d\n", num_pyramid_matrices, base_iter);
        // }

        // Fourth case
        p = 0;

        for (int i = 0; i < level - 1; i++)
        {
            int layer_items = (level - i) * (level - i - 1) / 2;
            for (int j = 0; j < level - i - 1; j++)
            {
                p++;
                for (int k = 1; k < level - i - j - 1; k++)
                {
                    int e[4] = {
                        p, 
                        p + layer_items + level - i - j - 1 + level - i - j - 1,
                        p + layer_items + level - i - j - 1,
                        p + level - i - j - 1
                    };

                    gathered[num_total_matrices * MATRIX_B_SIZE * macro_tet_idx + base_iter + vals_iter + 0] = localX[e[0] * BLOCK_SIZE + threadIdx.x];
                    gathered[num_total_matrices * MATRIX_B_SIZE * macro_tet_idx + base_iter + vals_iter + 8] = localX[e[1] * BLOCK_SIZE + threadIdx.x];
                    gathered[num_total_matrices * MATRIX_B_SIZE * macro_tet_idx + base_iter + vals_iter + 16] = localX[e[2] * BLOCK_SIZE + threadIdx.x];
                    gathered[num_total_matrices * MATRIX_B_SIZE * macro_tet_idx + base_iter + vals_iter + 24] = localX[e[3] * BLOCK_SIZE + threadIdx.x];

                    vals_iter += 1;
                    base_iter += (vals_iter / 8) * MATRIX_B_SIZE;
                    vals_iter = vals_iter % 8;

                    p++;
                }
                p++;
            }
            p++;
        }

        if (vals_iter > 0) base_iter += MATRIX_B_SIZE;
        vals_iter = 0;

        // if (macro_tet_idx == 0) {
        //     printf("tet: %d, base_iter A3 %d\n", num_pyramid_matrices, base_iter);
        // }

        // Fifth case
        p = 0;

        for (int i = 1; i < level - 1; i++)
        {
            p = p + level - i + 1;
            int layer_items = (level - i) * (level - i - 1) / 2;
            for (int j = 0; j < level - i - 1; j++)
            {
                p++;
                for (int k = 1; k < level - i - j - 1; k++)
                {
                    int e[4] = {
                        p, 
                        p + layer_items + level - i - j + level - i,
                        p + layer_items + level - i,
                        p + layer_items + level - i - j + level - i - 1
                    };

                    gathered[num_total_matrices * MATRIX_B_SIZE * macro_tet_idx + base_iter + vals_iter + 0] = localX[e[0] * BLOCK_SIZE + threadIdx.x];
                    gathered[num_total_matrices * MATRIX_B_SIZE * macro_tet_idx + base_iter + vals_iter + 8] = localX[e[1] * BLOCK_SIZE + threadIdx.x];
                    gathered[num_total_matrices * MATRIX_B_SIZE * macro_tet_idx + base_iter + vals_iter + 16] = localX[e[2] * BLOCK_SIZE + threadIdx.x];
                    gathered[num_total_matrices * MATRIX_B_SIZE * macro_tet_idx + base_iter + vals_iter + 24] = localX[e[3] * BLOCK_SIZE + threadIdx.x];

                    vals_iter += 1;
                    base_iter += (vals_iter / 8) * MATRIX_B_SIZE;
                    vals_iter = vals_iter % 8;

                    p++;
                }
                p++;
            }
            p++;
        }

        if (vals_iter > 0) base_iter += MATRIX_B_SIZE;
        vals_iter = 0;

        // if (macro_tet_idx == 0) {
        //     printf("tet: %d, base_iter A4 %d\n", num_pyramid_matrices, base_iter);
        // }

        // Sixth case
        p = 0;
        for (int i = 0; i < level - 1; i++)
        {
            int layer_items = (level - i) * (level - i - 1) / 2;
            for (int j = 0; j < level - i - 1; j++)
            {
                p++;
                for (int k = 1; k < level - i - j - 1; k++)
                {
                    int e[4] = {
                        p, 
                        p + level - i - j,
                        p + layer_items + level - i - j - 1 + level - i - j - 1,
                        p + level - i - j - 1
                    };

                    gathered[num_total_matrices * MATRIX_B_SIZE * macro_tet_idx + base_iter + vals_iter + 0] = localX[e[0] * BLOCK_SIZE + threadIdx.x];
                    gathered[num_total_matrices * MATRIX_B_SIZE * macro_tet_idx + base_iter + vals_iter + 8] = localX[e[1] * BLOCK_SIZE + threadIdx.x];
                    gathered[num_total_matrices * MATRIX_B_SIZE * macro_tet_idx + base_iter + vals_iter + 16] = localX[e[2] * BLOCK_SIZE + threadIdx.x];
                    gathered[num_total_matrices * MATRIX_B_SIZE * macro_tet_idx + base_iter + vals_iter + 24] = localX[e[3] * BLOCK_SIZE + threadIdx.x];

                    vals_iter += 1;
                    base_iter += (vals_iter / 8) * MATRIX_B_SIZE;
                    vals_iter = vals_iter % 8;

                    p++;
                }
                p++;
            }
            p++;
        }

        // if (macro_tet_idx == 0) {
        //     printf("tet: %d, base_iter A5 %d\n", num_inverted_matrices, base_iter);
        // }

        // enter collective operations where everything is synchronized
        int max_macro_tet_idx = blockIdx.x * blockDim.x + BLOCK_SIZE;
        if (max_macro_tet_idx > n_macro_tets) max_macro_tet_idx = n_macro_tets;

        if (macro_tet_idx == 0) {
            printf("num_total_matrices: %d\n", num_total_matrices);
        }

        // for (int local_macro_tet_idx = 0; local_macro_tet_idx < BLOCK_SIZE; local_macro_tet_idx += 1) {
        for (int local_macro_tet_idx = blockIdx.x * blockDim.x; local_macro_tet_idx < max_macro_tet_idx; local_macro_tet_idx += 1) {

            int num_micro_tet_matrices[6] = {num_corner_matrices, num_pyramid_matrices, num_pyramid_matrices, 
                num_pyramid_matrices, num_inverted_matrices, num_pyramid_matrices };

            int category_start_idx = 0;

            for (int category_idx = 0; category_idx < 6; category_idx += 1) {
                wmma::load_matrix_sync(a_frag, &microL[category_idx * MATRIX_A_SIZE + 6 * MATRIX_A_SIZE * local_macro_tet_idx], 4);
                for (int matrix_idx = category_start_idx; matrix_idx < category_start_idx + num_micro_tet_matrices[category_idx]; matrix_idx += 1) {
                // for (int matrix_idx = 0; matrix_idx < num_micro_tet_matrices[category_idx]; matrix_idx += 1) {

                    wmma::load_matrix_sync(b_frag, &gathered[num_total_matrices * MATRIX_B_SIZE * local_macro_tet_idx + matrix_idx * MATRIX_B_SIZE], 8);

                    // Perform the matrix multiplication
                    wmma::mma_sync(c_frag, a_frag, b_frag, c_frag);

                    // printf("writing to matrix idx %d, location %d\n", matrix_idx, num_total_matrices * MATRIX_C_SHRINKED * local_macro_tet_idx + matrix_idx * MATRIX_C_SHRINKED);

                    // Store the output
                    wmma::store_matrix_sync(&to_scatter[num_total_matrices * MATRIX_C_SHRINKED * local_macro_tet_idx + matrix_idx * MATRIX_C_SHRINKED], c_frag, 8, wmma::mem_row_major);
                }
                category_start_idx += num_micro_tet_matrices[category_idx];
            }
        }

        // go back to everyday single threading mood
        vals_iter = 0;
        base_iter = 0;

        p = 0;
        for (int i = 0; i < level - 1; i++)
        {
            int layer_items = (level - i + 1) * (level - i) / 2;
            for (int j = 0; j < level - i - 1; j++)
            {
                for (int k = 0; k < level - i - j - 1; k++)
                {
                    int e[4] = {p, p + layer_items - j, p + level - i - j, p + 1};

                    localY[e[0] * BLOCK_SIZE + threadIdx.x] += to_scatter[num_total_matrices * MATRIX_C_SHRINKED * macro_tet_idx + base_iter + vals_iter + 0];
                    localY[e[1] * BLOCK_SIZE + threadIdx.x] += to_scatter[num_total_matrices * MATRIX_C_SHRINKED * macro_tet_idx + base_iter + vals_iter + 8];
                    localY[e[2] * BLOCK_SIZE + threadIdx.x] += to_scatter[num_total_matrices * MATRIX_C_SHRINKED * macro_tet_idx + base_iter + vals_iter + 16];
                    localY[e[3] * BLOCK_SIZE + threadIdx.x] += to_scatter[num_total_matrices * MATRIX_C_SHRINKED * macro_tet_idx + base_iter + vals_iter + 24];

                    if (macro_tet_idx == 0) {
                        printf("First C: [%lu] [%lu] [%lu] [%lu]\n", 
                            num_total_matrices * MATRIX_C_SHRINKED * macro_tet_idx + base_iter + vals_iter + 0, 
                            (num_total_matrices * MATRIX_C_SHRINKED * macro_tet_idx) + base_iter + vals_iter + 8,
                            num_total_matrices * MATRIX_C_SHRINKED * macro_tet_idx + base_iter + vals_iter + 16,
                            (num_total_matrices * MATRIX_C_SHRINKED * macro_tet_idx) + base_iter + vals_iter + 24);
                        // printf("First C: %lf %lf %lf %lf\n", localY[e[0] * BLOCK_SIZE + threadIdx.x], localY[e[1] * BLOCK_SIZE + threadIdx.x], 
                        //     localY[e[2] * BLOCK_SIZE + threadIdx.x], localY[e[3] * BLOCK_SIZE + threadIdx.x]);
                        // printf("First: %d %d %d %d\n", e[0], e[1], e[2], e[3]);
                        printf("vals_iter: %d, base_iter: %d\n", vals_iter, base_iter);
                    }

                    vals_iter += 1;
                    base_iter += (vals_iter / 8) * MATRIX_C_SHRINKED;
                    vals_iter = vals_iter % 8;

                    // printf("First: %d %d %d %d\n", e0, e3, e2, e1);
                    p++;
                }
                p++;
            }
            p++;
        }

        if (vals_iter > 0) base_iter += MATRIX_C_SHRINKED;
        vals_iter = 0;

        if (macro_tet_idx == 0) {
            printf("tet: %d, base_iter C0 %d\n", num_corner_matrices, base_iter);
            for (int assembled_mat_idx = 0; assembled_mat_idx < base_iter / MATRIX_C_SHRINKED; assembled_mat_idx += 1) {
                printf("\nSystem C0 Matrix %d\n", assembled_mat_idx);
                print_matrix(&to_scatter[num_total_matrices * MATRIX_C_SHRINKED * macro_tet_idx + assembled_mat_idx * MATRIX_C_SHRINKED], 8, 8);
            }
        }

        // Second case
        p = 0;
        for (int i = 0; i < level - 1; i++)
        {
            int layer_items = (level - i) * (level - i - 1) / 2;
            for (int j = 0; j < level - i - 1; j++)
            {
                p++;
                for (int k = 1; k < level - i - j - 1; k++)
                {
                    int e[4] = {
                        p, 
                        p + layer_items + level - i - j - 1 + level - i - j - 1,
                        p + layer_items + level - i - j, 
                        p + layer_items + level - i - j - 1
                    };

                    localY[e[0] * BLOCK_SIZE + threadIdx.x] += to_scatter[num_total_matrices * MATRIX_C_SHRINKED * macro_tet_idx + base_iter + vals_iter + 0];
                    localY[e[1] * BLOCK_SIZE + threadIdx.x] += to_scatter[num_total_matrices * MATRIX_C_SHRINKED * macro_tet_idx + base_iter + vals_iter + 8];
                    localY[e[2] * BLOCK_SIZE + threadIdx.x] += to_scatter[num_total_matrices * MATRIX_C_SHRINKED * macro_tet_idx + base_iter + vals_iter + 16];
                    localY[e[3] * BLOCK_SIZE + threadIdx.x] += to_scatter[num_total_matrices * MATRIX_C_SHRINKED * macro_tet_idx + base_iter + vals_iter + 24];

                    vals_iter += 1;
                    base_iter += (vals_iter / 8) * MATRIX_C_SHRINKED;
                    vals_iter = vals_iter % 8;

                    p++;
                }
                p++;
            }
            p++;
        }

        if (vals_iter > 0) base_iter += MATRIX_C_SHRINKED;
        vals_iter = 0;

        if (macro_tet_idx == 0) {
            printf("tet: %d, base_iter C1 %d\n", num_pyramid_matrices, base_iter);
            for (int assembled_mat_idx = num_corner_matrices; assembled_mat_idx < base_iter / MATRIX_C_SHRINKED; assembled_mat_idx += 1) {
                printf("\nSystem C1 Matrix %d\n", assembled_mat_idx);
                print_matrix(&to_scatter[num_total_matrices * MATRIX_C_SHRINKED * macro_tet_idx + assembled_mat_idx * MATRIX_C_SHRINKED], 8, 8);
            }
        }

        // Third case
        p = 0;

        for (int i = 0; i < level - 1; i++)
        {
            int layer_items = (level - i) * (level - i - 1) / 2;
            for (int j = 0; j < level - i - 1; j++)
            {
                p++;
                for (int k = 1; k < level - i - j - 1; k++)
                {
                    int e[4] = {
                        p, 
                        p + layer_items + level - i - j,
                        p + layer_items + level - i - j - 1 + level - i - j - 1,
                        p + level - i - j
                    };

                    localY[e[0] * BLOCK_SIZE + threadIdx.x] += to_scatter[num_total_matrices * MATRIX_C_SHRINKED * macro_tet_idx + base_iter + vals_iter + 0];
                    localY[e[1] * BLOCK_SIZE + threadIdx.x] += to_scatter[num_total_matrices * MATRIX_C_SHRINKED * macro_tet_idx + base_iter + vals_iter + 8];
                    localY[e[2] * BLOCK_SIZE + threadIdx.x] += to_scatter[num_total_matrices * MATRIX_C_SHRINKED * macro_tet_idx + base_iter + vals_iter + 16];
                    localY[e[3] * BLOCK_SIZE + threadIdx.x] += to_scatter[num_total_matrices * MATRIX_C_SHRINKED * macro_tet_idx + base_iter + vals_iter + 24];

                    vals_iter += 1;
                    base_iter += (vals_iter / 8) * MATRIX_C_SHRINKED;
                    vals_iter = vals_iter % 8;

                    p++;
                }
                p++;
            }
            p++;
        }

        if (vals_iter > 0) base_iter += MATRIX_C_SHRINKED;
        vals_iter = 0;

        if (macro_tet_idx == 0) {
            printf("tet: %d, base_iter C2 %d\n", num_pyramid_matrices, base_iter);
            for (int assembled_mat_idx = num_corner_matrices + num_pyramid_matrices; assembled_mat_idx < base_iter / MATRIX_C_SHRINKED; assembled_mat_idx += 1) {
                printf("\nSystem C2 Matrix %d\n", assembled_mat_idx);
                print_matrix(&to_scatter[num_total_matrices * MATRIX_C_SHRINKED * macro_tet_idx + assembled_mat_idx * MATRIX_C_SHRINKED], 8, 8);
            }
        }

        // Fourth case
        p = 0;

        for (int i = 0; i < level - 1; i++)
        {
            int layer_items = (level - i) * (level - i - 1) / 2;
            for (int j = 0; j < level - i - 1; j++)
            {
                p++;
                for (int k = 1; k < level - i - j - 1; k++)
                {
                    int e[4] = {
                        p, 
                        p + layer_items + level - i - j - 1 + level - i - j - 1,
                        p + layer_items + level - i - j - 1,
                        p + level - i - j - 1
                    };

                    localY[e[0] * BLOCK_SIZE + threadIdx.x] += to_scatter[num_total_matrices * MATRIX_C_SHRINKED * macro_tet_idx + base_iter + vals_iter + 0];
                    localY[e[1] * BLOCK_SIZE + threadIdx.x] += to_scatter[num_total_matrices * MATRIX_C_SHRINKED * macro_tet_idx + base_iter + vals_iter + 8];
                    localY[e[2] * BLOCK_SIZE + threadIdx.x] += to_scatter[num_total_matrices * MATRIX_C_SHRINKED * macro_tet_idx + base_iter + vals_iter + 16];
                    localY[e[3] * BLOCK_SIZE + threadIdx.x] += to_scatter[num_total_matrices * MATRIX_C_SHRINKED * macro_tet_idx + base_iter + vals_iter + 24];

                    vals_iter += 1;
                    base_iter += (vals_iter / 8) * MATRIX_C_SHRINKED;
                    vals_iter = vals_iter % 8;

                    p++;
                }
                p++;
            }
            p++;
        }

        if (vals_iter > 0) base_iter += MATRIX_C_SHRINKED;
        vals_iter = 0;

        if (macro_tet_idx == 0) {
            printf("tet: %d, base_iter C3 %d\n", num_pyramid_matrices, base_iter);
            for (int assembled_mat_idx = num_corner_matrices + 2 * num_pyramid_matrices; assembled_mat_idx < base_iter / MATRIX_C_SHRINKED; assembled_mat_idx += 1) {
                printf("\nSystem C3 Matrix %d\n", assembled_mat_idx);
                print_matrix(&to_scatter[num_total_matrices * MATRIX_C_SHRINKED * macro_tet_idx + assembled_mat_idx * MATRIX_C_SHRINKED], 8, 8);
            }
        }

        // Fifth case
        p = 0;

        for (int i = 1; i < level - 1; i++)
        {
            p = p + level - i + 1;
            int layer_items = (level - i) * (level - i - 1) / 2;
            for (int j = 0; j < level - i - 1; j++)
            {
                p++;
                for (int k = 1; k < level - i - j - 1; k++)
                {
                    int e[4] = {
                        p, 
                        p + layer_items + level - i - j + level - i,
                        p + layer_items + level - i,
                        p + layer_items + level - i - j + level - i - 1
                    };

                    localY[e[0] * BLOCK_SIZE + threadIdx.x] += to_scatter[num_total_matrices * MATRIX_C_SHRINKED * macro_tet_idx + base_iter + vals_iter + 0];
                    localY[e[1] * BLOCK_SIZE + threadIdx.x] += to_scatter[num_total_matrices * MATRIX_C_SHRINKED * macro_tet_idx + base_iter + vals_iter + 8];
                    localY[e[2] * BLOCK_SIZE + threadIdx.x] += to_scatter[num_total_matrices * MATRIX_C_SHRINKED * macro_tet_idx + base_iter + vals_iter + 16];
                    localY[e[3] * BLOCK_SIZE + threadIdx.x] += to_scatter[num_total_matrices * MATRIX_C_SHRINKED * macro_tet_idx + base_iter + vals_iter + 24];

                    vals_iter += 1;
                    base_iter += (vals_iter / 8) * MATRIX_C_SHRINKED;
                    vals_iter = vals_iter % 8;

                    p++;
                }
                p++;
            }
            p++;
        }

        if (vals_iter > 0) base_iter += MATRIX_C_SHRINKED;
        vals_iter = 0;

        if (macro_tet_idx == 0) {
            printf("tet: %d, base_iter C4 %d\n", num_inverted_matrices, base_iter);
            for (int assembled_mat_idx = num_corner_matrices + 3 * num_pyramid_matrices; assembled_mat_idx < base_iter / MATRIX_C_SHRINKED; assembled_mat_idx += 1) {
                printf("\nSystem C4 Matrix %d\n", assembled_mat_idx);
                print_matrix(&to_scatter[num_total_matrices * MATRIX_C_SHRINKED * macro_tet_idx + assembled_mat_idx * MATRIX_C_SHRINKED], 8, 8);
            }
        }

        // Sixth case
        p = 0;
        for (int i = 0; i < level - 1; i++)
        {
            int layer_items = (level - i) * (level - i - 1) / 2;
            for (int j = 0; j < level - i - 1; j++)
            {
                p++;
                for (int k = 1; k < level - i - j - 1; k++)
                {
                    int e[4] = {
                        p, 
                        p + level - i - j,
                        p + layer_items + level - i - j - 1 + level - i - j - 1,
                        p + level - i - j - 1
                    };

                    localY[e[0] * BLOCK_SIZE + threadIdx.x] += to_scatter[num_total_matrices * MATRIX_C_SHRINKED * macro_tet_idx + base_iter + vals_iter + 0];
                    localY[e[1] * BLOCK_SIZE + threadIdx.x] += to_scatter[num_total_matrices * MATRIX_C_SHRINKED * macro_tet_idx + base_iter + vals_iter + 8];
                    localY[e[2] * BLOCK_SIZE + threadIdx.x] += to_scatter[num_total_matrices * MATRIX_C_SHRINKED * macro_tet_idx + base_iter + vals_iter + 16];
                    localY[e[3] * BLOCK_SIZE + threadIdx.x] += to_scatter[num_total_matrices * MATRIX_C_SHRINKED * macro_tet_idx + base_iter + vals_iter + 24];

                    vals_iter += 1;
                    base_iter += (vals_iter / 8) * MATRIX_C_SHRINKED;
                    vals_iter = vals_iter % 8;

                    p++;
                }
                p++;
            }
            p++;
        }

        if (macro_tet_idx == 0) {
            printf("tet: %d, base_iter C5 %d = total %d\n", num_pyramid_matrices, base_iter, num_total_matrices);
            for (int assembled_mat_idx = num_corner_matrices + 3 * num_pyramid_matrices + num_inverted_matrices; assembled_mat_idx < base_iter / MATRIX_C_SHRINKED; assembled_mat_idx += 1) {
                printf("\nSystem C5 Matrix %d\n", assembled_mat_idx);
                print_matrix(&to_scatter[num_total_matrices * MATRIX_C_SHRINKED * macro_tet_idx + assembled_mat_idx * MATRIX_C_SHRINKED], 8, 8);
            }
        }

        for (int micro_node_idx = 0; micro_node_idx < n_micro_nodes; micro_node_idx++) {
            vecY[micro_node_idx * stride + macro_tet_idx] = localY[micro_node_idx * BLOCK_SIZE + threadIdx.x];

            if (macro_tet_idx == 0) {
                printf("%lf ", localY[micro_node_idx * BLOCK_SIZE + threadIdx.x]);
            }
        }
            if (macro_tet_idx == 0) {
                printf("\n");
            }
    }
}

// Kernel to apply Dirichlet boundary conditions
__global__ void applyDirichlet(real_t *Ax, real_t *rhs, size_t num_macro_tets, size_t stride, size_t *dirichlet_nodes, size_t num_dirichlet_nodes) {
    for (size_t idx = blockIdx.x * blockDim.x + threadIdx.x; idx < num_macro_tets;
         idx += blockDim.x * gridDim.x) {
            for (int j = 0; j < num_dirichlet_nodes; j += 1) {
                size_t dirichlet_node_idx = dirichlet_nodes[j];
                Ax[dirichlet_node_idx * stride + idx] = rhs[dirichlet_node_idx * stride + idx];
            }
    }
}

// Kernel to compute the residual r = rhs - Ax
__global__ void computeResidual(real_t *r, real_t *rhs, real_t *Ax, size_t num_macro_tets, size_t stride, size_t num_local_nodes) {
    // iterate over some tetrahedrons
    for (size_t macro_tet_idx = blockIdx.x * blockDim.x + threadIdx.x; macro_tet_idx < num_macro_tets;
         macro_tet_idx += blockDim.x * gridDim.x) {
            // iterate over the local nodes
            for (size_t node_idx = 0; node_idx < num_local_nodes; node_idx += 1) {
                r[node_idx * stride + macro_tet_idx] = rhs[node_idx * stride + macro_tet_idx] - Ax[node_idx * stride + macro_tet_idx];
            }
    }
}

// Kernel for vector dot product: result = sum(a[i] * b[i])
__global__ void dotProduct(const real_t* a, const real_t* b, real_t* result, size_t num_macro_tets, size_t stride, size_t num_local_nodes) {
    for (size_t macro_tet_idx = blockIdx.x * blockDim.x + threadIdx.x; macro_tet_idx < num_macro_tets;
         macro_tet_idx += blockDim.x * gridDim.x) {
            // iterate over the local nodes
            for (size_t node_idx = 0; node_idx < num_local_nodes; node_idx += 1) {
                result[macro_tet_idx] += a[node_idx * stride + macro_tet_idx] * b[node_idx * stride + macro_tet_idx];
            }
            if (macro_tet_idx == 0) {
                printf("dotProduct of %d: %lf\n", macro_tet_idx, result[macro_tet_idx]);
            }
    }
}

// Kernel for vector update: y = alpha * x + b
__global__ void vectorAdd(real_t *y, const real_t *alpha, const real_t *x, const real_t *b, size_t stride, size_t num_macro_tets, size_t num_local_nodes) {
    // iterate over some tetrahedrons
    for (size_t macro_tet_idx = blockIdx.x * blockDim.x + threadIdx.x; macro_tet_idx < num_macro_tets;
         macro_tet_idx += blockDim.x * gridDim.x) {
            // iterate over the local nodes
            for (size_t node_idx = 0; node_idx < num_local_nodes; node_idx += 1) {
                y[node_idx * stride + macro_tet_idx] = alpha[macro_tet_idx] * x[node_idx * stride + macro_tet_idx] + b[node_idx * stride + macro_tet_idx];
            }

            if (macro_tet_idx == 0) {
                printf("vecX after vectorAdd: \n");
                for (int n = 0; n < 100; n += 1) {
                    printf("%lf ", y[n * stride + macro_tet_idx]);
                }
                printf("\n");
            }
    }

}

// Kernel for vector update: x += alpha * r 
__global__ void vectorUpdate(real_t *x, const real_t alpha, const real_t *r, size_t stride, size_t num_macro_tets, size_t num_local_nodes) {
    // iterate over some tetrahedrons
    for (size_t macro_tet_idx = blockIdx.x * blockDim.x + threadIdx.x; macro_tet_idx < num_macro_tets;
         macro_tet_idx += blockDim.x * gridDim.x) {
            // iterate over the local nodes
            for (size_t node_idx = 0; node_idx < num_local_nodes; node_idx += 1) {
                x[node_idx * stride + macro_tet_idx] = alpha * r[node_idx * stride + macro_tet_idx];
            }
    }

}

// Kernel for vector update: x = x - alpha * p
__global__ void vectorMinus(real_t* x, const real_t* p, real_t *alpha, size_t stride, size_t num_macro_tets, size_t num_local_nodes) {
    // iterate over some tetrahedrons
    for (size_t macro_tet_idx = blockIdx.x * blockDim.x + threadIdx.x; macro_tet_idx < num_macro_tets;
         macro_tet_idx += blockDim.x * gridDim.x) {
            // iterate over the local nodes
            for (size_t node_idx = 0; node_idx < num_local_nodes; node_idx += 1) {
                x[node_idx * stride + macro_tet_idx] -= alpha[macro_tet_idx] * p[node_idx * stride + macro_tet_idx];
            }

            if (macro_tet_idx == 0) {
                printf("p in vectorMinus: \n");
                for (int n = 0; n < 100; n += 1) {
                    printf("%lf ", p[n * stride + macro_tet_idx]);
                }
                printf("\n");
                printf("alpha: %lf\n", alpha[macro_tet_idx]);
            }
    }

}

// Kernel for division update: alpha = up / down
__global__ void scalarDivide(real_t* alpha, const real_t* up, real_t *down, size_t num_macro_tets) {
    // iterate over some tetrahedrons
    for (size_t macro_tet_idx = blockIdx.x * blockDim.x + threadIdx.x; macro_tet_idx < num_macro_tets;
         macro_tet_idx += blockDim.x * gridDim.x) {
            // iterate over the local nodes
            alpha[macro_tet_idx] = up[macro_tet_idx] / down[macro_tet_idx];
            if (macro_tet_idx == 0) {
                printf("scalarDivide of %lf/%lf: %lf\n", up[macro_tet_idx], down[macro_tet_idx], alpha[macro_tet_idx]);
            }
    }

}

__global__ void checkConvergence(const real_t tol, const real_t* residual, int num_macro_tets, size_t* converged) {
    for (size_t macro_tet_idx = blockIdx.x * blockDim.x + threadIdx.x; macro_tet_idx < num_macro_tets;
         macro_tet_idx += blockDim.x * gridDim.x) {
        if (residual[macro_tet_idx] >= tol * tol) {
            *converged = 0;
            return;
        }
    }
}

// CUDA Kernel to set the Dirichlet boundary conditions
__global__ void setDirichletBoundaryConditions(size_t *dirichlet_nodes, real_t *rhs, real_t *x, size_t num_macro_tets, size_t stride, real_t *dirichlet_values, size_t num_dirichlet_nodes) {
    for (size_t macro_tet_idx = blockIdx.x * blockDim.x + threadIdx.x; macro_tet_idx < num_macro_tets;
         macro_tet_idx += blockDim.x * gridDim.x) {
            // iterate over the local nodes
            for (size_t i = 0; i < num_dirichlet_nodes; i += 1) {
                size_t local_node_idx = dirichlet_nodes[i];
                rhs[local_node_idx * stride + macro_tet_idx] = dirichlet_values[i];
                x[local_node_idx * stride + macro_tet_idx] = dirichlet_values[i];
            }
    }
}

void set_boundary_conditions_cuda(size_t num_nodes, real_t *rhs, real_t *x, size_t num_macro_tets, size_t stride, size_t **dirichlet_nodes, size_t *num_dirichlet_nodes)
{
    *num_dirichlet_nodes = 2;
    checkCudaError(hipMalloc(dirichlet_nodes, (*num_dirichlet_nodes) * sizeof(size_t)));

    // Set the Dirichlet nodes (macro_tet_idx.g., first and last nodes)
    size_t h_dirichlet_nodes[] = {0, num_nodes - 1};
    checkCudaError(hipMemcpy(*dirichlet_nodes, h_dirichlet_nodes, (*num_dirichlet_nodes) * sizeof(size_t), hipMemcpyHostToDevice));

    // Set the Dirichlet values corresponding to the Dirichlet nodes
    real_t h_dirichlet_values[] = {1.0, 0.0};

    real_t *d_dirichlet_values;
    checkCudaError(hipMalloc(&d_dirichlet_values, (*num_dirichlet_nodes) * sizeof(real_t)));
    checkCudaError(hipMemcpy(d_dirichlet_values, h_dirichlet_values, (*num_dirichlet_nodes) * sizeof(real_t), hipMemcpyHostToDevice));

    // Launch the kernel to set the Dirichlet boundary conditions
    int threadsPerBlock = BLOCK_SIZE;
    int numBlocks = (num_macro_tets + threadsPerBlock - 1) / threadsPerBlock;
    setDirichletBoundaryConditions<<<numBlocks, threadsPerBlock>>>(*dirichlet_nodes, rhs, x, num_macro_tets, stride, d_dirichlet_values, *num_dirichlet_nodes);

    ifLastErrorExists("Kernel launch failed");

    // Free the temporary device memory for Dirichlet values
    checkCudaError(hipFree(d_dirichlet_values));
}

__host__ real_t *solve_using_gradient_descent(int tetra_level, int num_macro_tets, int num_nodes, real_t *macro_jacobians)
{
    // Allocate variables for boundary conditions
    int max_iter = 5;
    real_t tol = 1e-2;
    real_t *h_x, *h_r;
    checkCudaError(hipHostMalloc(&h_x, num_macro_tets * sizeof(real_t) * num_nodes));
    checkCudaError(hipHostMalloc(&h_r, num_macro_tets * sizeof(real_t) * num_nodes));

    // Allocate GPU memory
    real_t *d_b, *d_x, *d_r, *d_Ax;
    checkCudaError(hipMalloc(&d_b, num_macro_tets * num_nodes * sizeof(real_t)));
    checkCudaError(hipMalloc(&d_x, num_macro_tets * num_nodes * sizeof(real_t)));
    checkCudaError(hipMalloc(&d_Ax, num_macro_tets * num_nodes * sizeof(real_t)));
    checkCudaError(hipMalloc(&d_r, num_macro_tets * num_nodes * sizeof(real_t)));

    hipblasHandle_t cublas_handle;
    hipblasCreate(&cublas_handle);

    size_t *d_dirichlet_nodes;
    size_t num_dirichlet_nodes;

    int stride = num_macro_tets;

    set_boundary_conditions_cuda(num_nodes, d_b, d_x, num_macro_tets, stride, &d_dirichlet_nodes, &num_dirichlet_nodes);
    checkCudaError(hipMemcpy(h_x, d_x, sizeof(real_t) * num_macro_tets * num_nodes, hipMemcpyDeviceToHost));

    int threadsPerBlock = BLOCK_SIZE;
    int numBlocks = (num_macro_tets + threadsPerBlock - 1) / threadsPerBlock;

    const int MATRIX_B_SIZE = 4 * 8;
    const int MATRIX_A_SIZE = 8 * 4;
    const int MATRIX_C_SIZE = 8 * 8;

    const int MATRIX_C_SHRINKED = 8 * 4;

    int num_corner_matrices = (compute_tets_number_of_category(tetra_level, 0) + 7) / 8;
    int num_pyramid_matrices = (compute_tets_number_of_category(tetra_level, 1) + 7) / 8;
    int num_inverted_matrices = (compute_tets_number_of_category(tetra_level, 4) + 7) / 8;

    int n_micro_nodes = compute_nodes_number(tetra_level);
    int num_micro_tets = compute_tets_number(tetra_level);

    int num_total_matrices = num_corner_matrices + num_pyramid_matrices * 4 + num_inverted_matrices;

    real_t *d_matA, *d_matC, *d_matB;
    checkCudaError(hipMalloc(&d_matA, num_macro_tets * MATRIX_A_SIZE * 6 * sizeof(real_t)));
    checkCudaError(hipMalloc(&d_matB, num_macro_tets * MATRIX_B_SIZE * num_total_matrices * sizeof(real_t)));
    checkCudaError(hipMalloc(&d_matC, num_macro_tets * MATRIX_C_SIZE * num_total_matrices * sizeof(real_t)));

    checkCudaError(hipMemset(d_matA, num_macro_tets * MATRIX_A_SIZE * 6 * sizeof(real_t), 0));
    checkCudaError(hipMemset(d_matC, num_macro_tets * MATRIX_C_SIZE * num_total_matrices * sizeof(real_t), 0));

    printf("host num_total_matrices: %d allocated: %d\n", num_total_matrices, num_macro_tets * MATRIX_C_SIZE * num_total_matrices);

    int sharedMemoryBytes = 100000; // 100KB (100000, tetra_level=4) or 164KB (164000, tetra_level=5)
    int requiredBytes = sizeof(real_t) * BLOCK_SIZE * (n_micro_nodes * 2);
    checkCudaError(hipFuncSetAttribute(reinterpret_cast<const void*>(cu_macro_tet4_laplacian_apply_kernel), hipFuncAttributeMaxDynamicSharedMemorySize, sharedMemoryBytes));
    printf("Bytes requested: %d, needed: %d\n", sharedMemoryBytes, requiredBytes);

    // Start Gradient Descent iterations
    int iter = 0;
    real_t gamma = 7 * 1e-1;
    while (iter < max_iter) {

        // Initialize r = b - A * x
        cu_macro_tet4_laplacian_apply_kernel<<<numBlocks, threadsPerBlock, requiredBytes>>>(num_macro_tets, num_micro_tets, num_nodes, 
            num_macro_tets, tetra_level, macro_jacobians, d_matA, d_matB, d_matC, d_x, d_Ax);
        ifLastErrorExists("Kernel launch failed");

        applyDirichlet<<<numBlocks, threadsPerBlock>>>(d_Ax, d_b, num_macro_tets, stride, d_dirichlet_nodes, num_dirichlet_nodes);
        ifLastErrorExists("Kernel launch failed");

        computeResidual<<<numBlocks, threadsPerBlock>>>(d_r, d_b, d_Ax, num_macro_tets, stride, num_nodes);
        ifLastErrorExists("Kernel launch failed");

        // cuBLAS for reduction
        // minSquareError computeNorm
        real_t norm_r = 0;
        if (sizeof(real_t) == 4) {
            checkCUBLASError(hipblasSnrm2(cublas_handle, num_macro_tets * num_nodes, (float *) d_r, 1, (float *) &norm_r));
        } else if (sizeof(real_t) == 8) {
            checkCUBLASError(hipblasDnrm2(cublas_handle, num_macro_tets * num_nodes, (double *) d_r, 1, (double *) &norm_r));
        }
        ifLastErrorExists("Kernel launch failed");

        printf("Iteration: %d, Global 2-norm = %lf\n", iter, norm_r);

        // Check for convergence
        if (norm_r < tol) {
            checkCudaError(hipMemcpy(&h_x, d_x, sizeof(real_t) * num_nodes * num_macro_tets, hipMemcpyDeviceToHost));
            for (int n = 0; n < num_nodes * num_macro_tets; n += num_macro_tets) {
                printf("%lf ", h_x[n]);
            }
            printf("Converged after %d iterations, 2-norm: %lf\n", iter, norm_r);
            // hipFree(converged);
            break;
        }

        // Update x = x + alpha * p
        vectorUpdate<<<numBlocks, threadsPerBlock>>>(d_x, gamma, d_r, stride, num_macro_tets, num_nodes);
        ifLastErrorExists("Kernel launch failed");
        
        // checkCudaError(hipMemcpy(h_x, d_x, sizeof(real_t) * num_macro_tets * num_nodes, hipMemcpyDeviceToHost));
        // printf("resulting x from vectorAdd: \n");
        // for (int n = 0; n < num_nodes * num_macro_tets; n += num_macro_tets) {
        //     printf("%lf ", h_x[n]);
        // }
        // printf("\n");

        iter++;
    }

    // Free GPU memory
    checkCudaError(hipFree(d_b));
    checkCudaError(hipFree(d_x));
    checkCudaError(hipFree(d_r));
    checkCudaError(hipFree(d_Ax));

    checkCudaError(hipFree(d_matA));
    checkCudaError(hipFree(d_matB));
    checkCudaError(hipFree(d_matC));

    // Free allocated memory
    checkCudaError(hipFree(d_dirichlet_nodes));

    checkCudaError(hipHostFree(h_r));

    hipblasDestroy(cublas_handle);

    return h_x;
}

void compute_A(real_t *p0, real_t *p1, real_t *p2, real_t *p3, real_t *A)
{
    for (int i = 0; i < 3; i++)
    {
        A[i] = p1[i] - p0[i];
        A[3 + i] = p2[i] - p0[i];
        A[6 + i] = p3[i] - p0[i];
    }
    // assert(determinant_3x3(A) > 0);
}

int main(void) {
    int tetra_level = 4;

    // Compute the number of nodes
    int num_nodes = compute_nodes_number(tetra_level);
    int num_micro_tets = compute_tets_number(tetra_level);

    int num_macro_tets = 320000;

    real_t *macro_jacobians, *h_macro_jacobians;
    checkCudaError(hipHostMalloc(&h_macro_jacobians, sizeof(real_t) * 9 * num_macro_tets));
    checkCudaError(hipMalloc(&macro_jacobians, sizeof(real_t) * 9 * num_macro_tets));

    for (int i = 0; i < num_macro_tets; i += 1) {
        real_t macro_J[9];
        real_t p0[3] = {0, 0, 0};
        real_t p1[3] = {1, 0, 0};
        real_t p2[3] = {0, 1, 0};
        real_t p3[3] = {0, 0, 1};
        compute_A(p0, p1, p2, p3, macro_J);
        for (int j = 0; j < 9; j += 1) {
            h_macro_jacobians[j * num_macro_tets + i] = macro_J[j];
        }
    }

    checkCudaError(hipMemcpy(macro_jacobians, h_macro_jacobians, 9 * sizeof(real_t) * num_macro_tets, hipMemcpyHostToDevice));
    real_t *h_x = solve_using_gradient_descent(tetra_level, num_macro_tets, num_nodes, macro_jacobians);
    checkCudaError(hipHostFree(h_x));

    checkCudaError(hipHostFree(h_macro_jacobians));
    checkCudaError(hipFree(macro_jacobians));
    // solve_using_gradient_descent(tetra_level, num_nodes, num_micro_tets, macro_J);

    return 0;

}