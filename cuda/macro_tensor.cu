#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include <math.h>
#include <unistd.h>
#include <assert.h>

#include <hip/hip_runtime.h>
#include <mma.h>
#include <hip/hip_runtime.h>
#include <hip/hip_cooperative_groups.h>
#include <hipblas.h>

// nvcc macro.cu --std=c++11 -o cargo -arch=sm_75 -g -G -lcublas
using namespace nvcuda;
using namespace cooperative_groups;

#define BLOCK_SIZE 128
#define BLOCK_Y_SIZE 8
typedef double real_t;

#define checkCudaError(call)                                                \
{                                                                           \
    hipError_t err = call;                                                 \
    if (err != hipSuccess)                                                 \
    {                                                                       \
        fprintf(stderr, "CUDA Error: %s (code: %d), at %s:%d\n",            \
                hipGetErrorString(err), err, __FILE__, __LINE__);          \
        exit(EXIT_FAILURE);                                                 \
    }                                                                       \
}

#define ifLastErrorExists(msg)                                         \
{                                                                      \
    hipError_t err = hipGetLastError();                              \
    if (err != hipSuccess)                                            \
    {                                                                  \
        fprintf(stderr, "CUDA Error: %s, at %s:%d - %s\n",             \
                msg, __FILE__, __LINE__, hipGetErrorString(err));     \
        exit(EXIT_FAILURE);                                            \
    }                                                                  \
}

__device__ void print_matrix(real_t *matrix, int rows, int cols)
{
    for (int i = 0; i < rows; i++)
    {
        for (int j = 0; j < cols; j++)
        {
            printf("%f ", matrix[i * cols + j]);
        }
        printf("\n");
    }
    printf("\n");
}

__device__ real_t determinant_3x3(real_t *m) {
    // computes the inverse of a matrix m
    double det = m[0*3+0] * (m[1*3+1] * m[2*3+2] - m[2*3+1] * m[1*3+2]) -
        m[0*3+1] * (m[1*3+0] * m[2*3+2] - m[1*3+2] * m[2*3+0]) +
        m[0*3+2] * (m[1*3+0] * m[2*3+1] - m[1*3+1] * m[2*3+0]);
    // print_matrix(m, 3, 3);
    // printf("det(m) = %lf\n", det);
    return det;
}

__device__ void inverse_3x3_T(real_t *m, real_t *m_inv)
{
    real_t det_inv = 1.0 / determinant_3x3(m);

    m_inv[0*3+0] = (m[1*3+1] * m[2*3+2] - m[2*3+1] * m[1*3+2]) * det_inv;
    m_inv[1*3+0] = (m[0*3+2] * m[2*3+1] - m[0*3+1] * m[2*3+2]) * det_inv;
    m_inv[2*3+0] = (m[0*3+1] * m[1*3+2] - m[0*3+2] * m[1*3+1]) * det_inv;
    m_inv[0*3+1] = (m[1*3+2] * m[2*3+0] - m[1*3+0] * m[2*3+2]) * det_inv;
    m_inv[1*3+1] = (m[0*3+0] * m[2*3+2] - m[0*3+2] * m[2*3+0]) * det_inv;
    m_inv[2*3+1] = (m[1*3+0] * m[0*3+2] - m[0*3+0] * m[1*3+2]) * det_inv;
    m_inv[0*3+2] = (m[1*3+0] * m[2*3+1] - m[2*3+0] * m[1*3+1]) * det_inv;
    m_inv[1*3+2] = (m[2*3+0] * m[0*3+1] - m[0*3+0] * m[2*3+1]) * det_inv;
    m_inv[2*3+2] = (m[0*3+0] * m[1*3+1] - m[1*3+0] * m[0*3+1]) * det_inv;
}

__device__ void jacobian_to_laplacian(real_t *macro_J, real_t *micro_L, int tetra_level, int category) {
    real_t J_inv_trans[9];
    real_t micro_J[9];
    const real_t grad_ref_phi[4][3] = {
        {-1, -1, -1},
        {1, 0, 0},
        {0, 1, 0},
        {0, 0, 1}
    };
    real_t grad_phi[4][3];

    // have to match the row/col order of compute_A
    real_t u[3] = {macro_J[0], macro_J[1], macro_J[2]};
    real_t v[3] = {macro_J[3], macro_J[4], macro_J[5]};
    real_t w[3] = {macro_J[6], macro_J[7], macro_J[8]};

    if (category == 0) {
        // [u | v | w]
        for (int i = 0; i < 3; i++) {
            for (int j = 0; j < 3; j++) {
                micro_J[i * 3 + j] = macro_J[i * 3 + j] / tetra_level;
            }
        }
        assert(determinant_3x3(micro_J) > 0);
    } else if (category == 1) {
        // [-u + w | w | -u + v + w]
        for (int i = 0; i < 3; i++) {
            micro_J[i * 3 + 0] = (-u[i] + w[i]) / tetra_level;
            micro_J[i * 3 + 1] = (w[i]) / tetra_level;
            micro_J[i * 3 + 2] = (-u[i] + v[i] + w[i]) / tetra_level;
        }
        assert(determinant_3x3(micro_J) > 0);
    } else if (category == 2) {
        // [v | -u + v + w | w]
        for (int i = 0; i < 3; i++) {
            micro_J[i * 3 + 0] = v[i] / tetra_level;
            micro_J[i * 3 + 1] = (-u[i] + v[i] + w[i]) / tetra_level;
            micro_J[i * 3 + 2] = (w[i]) / tetra_level;
        }
        assert(determinant_3x3(micro_J) > 0);
    } else if (category == 3) {
        // [-u + v | -u + w | -u + v + w]
        for (int i = 0; i < 3; i++) {
            micro_J[i * 3 + 0] = (-u[i] + v[i]) / tetra_level;
            micro_J[i * 3 + 1] = (-u[i] + w[i]) / tetra_level;
            micro_J[i * 3 + 2] = (-u[i] + v[i] + w[i]) / tetra_level;
        }
        assert(determinant_3x3(micro_J) > 0);
    } else if (category == 4) {
        // [-v + w | w | -u + w]
        for (int i = 0; i < 3; i++) {
            micro_J[i * 3 + 0] = (-v[i] + w[i]) / tetra_level;
            micro_J[i * 3 + 1] = (w[i]) / tetra_level;
            micro_J[i * 3 + 2] = (-u[i] + w[i]) / tetra_level;
        }
        assert(determinant_3x3(micro_J) > 0);
    } else if (category == 5) {
        // [-u + v | -u + v + w | v]
        for (int i = 0; i < 3; i++) {
            micro_J[i * 3 + 0] = (-u[i] + v[i]) / tetra_level;
            micro_J[i * 3 + 1] = (-u[i] + v[i] + w[i]) / tetra_level;
            micro_J[i * 3 + 2] = (v[i]) / tetra_level;
        }
        assert(determinant_3x3(micro_J) > 0);
    }

    inverse_3x3_T(micro_J, J_inv_trans);

    for (int i = 0; i < 4; i++) {
        for (int j = 0; j < 3; j++) {
            grad_phi[i][j] = 0;
            for (int k = 0; k < 3; k++) {
                grad_phi[i][j] += J_inv_trans[j * 3 + k] * grad_ref_phi[i][k];
            }
        }
    }

    for (int i = 0; i < 4; i++) {
        for (int j = 0; j < 4; j++) {
            real_t dot_product = 0;
            for (int k = 0; k < 3; k++) {
                dot_product += grad_phi[i][k] * grad_phi[j][k];
            }
            micro_L[i * 4 + j] = dot_product * determinant_3x3(micro_J) / 6.0;
        }
    }

}

template <typename real_t>
__global__ void cu_macro_tet4_laplacian_apply_kernel(
        const size_t nelements,
        const size_t stride,  // Stride here represents the number of macro-elements (aligned to 256 bytes?)
        int tetra_level, 
        const real_t *const macro_jacobians,
        const real_t *const vecX,
        real_t *const vecY) {

    int level = tetra_level + 1;

    real_t macro_J[9];
    real_t micro_L[32];

    __shared__ real_t results[BLOCK_Y_SIZE][64];
    __shared__ real_t vals_gathered[BLOCK_Y_SIZE][256];
    __shared__ real_t vals_to_scatter[BLOCK_Y_SIZE][256];

    // Declare the fragments
    wmma::fragment<wmma::matrix_a, 8, 8, 4, double, wmma::row_major> a_frag;
    wmma::fragment<wmma::matrix_b, 8, 8, 4, double, wmma::row_major> b_frag;
    wmma::fragment<wmma::accumulator, 8, 8, 4, double> c_frag;

    // Initialize the output to zero
    wmma::fill_fragment(c_frag, 0.0);

    for (size_t macro_idx = blockIdx.x * blockDim.x + threadIdx.y; macro_idx < nelements;
         macro_idx += blockDim.x * gridDim.x) {

// #pragma unroll(9)
        for (int d = 0; d < 9; d++) {
            macro_J[d] = macro_jacobians[d * stride + macro_idx];
        }

        jacobian_to_laplacian(macro_J, micro_L, tetra_level, 0);
        // wmma::load_matrix_sync(a_frag, micro_L, 4);

        int vals_iter = 0;
        int p = 0;

        if (threadIdx.x < 4) {
            for (int i = 0; i < level - 1; i++)
            {
                int layer_items = (level - i + 1) * (level - i) / 2;
                for (int j = 0; j < level - i - 1; j++)
                {
                    for (int k = 0; k < level - i - j - 1; k++)
                    {
                        int e[4] = {p, p + layer_items - j, p + level - i - j, p + 1};
                        vals_gathered[threadIdx.y][4 * vals_iter + threadIdx.x] = vecX[e[threadIdx.x] * stride + macro_idx];
                        vals_iter += 1;

                        // printf("First: %d %d %d %d\n", e[0], e[1], e[2], e[3]);

                        if (macro_idx == 0 && threadIdx.x == 0 && p < 2) {
                            printf("p:%d vals_gathered[%d, %d]: %lf\n", p, threadIdx.x, threadIdx.y, vals_gathered[threadIdx.y][4 * vals_iter + threadIdx.x]);
                        }

                        p++;
                    }
                    p++;
                }
                p++;
            }
        }

        __syncwarp();
        // TODO: think about the case where we don't have enough sub tetrahedrons (not divisible by 32)
        for (int i = 0; i < vals_iter; i += 32) {
            // Load the inputs
            // wmma::load_matrix_sync(b_frag, &vals_gathered[threadIdx.y][i], 8);
            // Perform the matrix multiplication
            // wmma::mma_sync(c_frag, a_frag, b_frag, c_frag);
            // Store the output (sync is necessary for &vals_to_scatter[i] due to padding overwritting memory)
            // wmma::store_matrix_sync(results[threadIdx.y], c_frag, 8, // wmma::mem_row_major);
            if (threadIdx.x < 4) {
                for (int j = 0; j < 8; j += 1) {
                    vals_to_scatter[threadIdx.y][i + j * 4 + threadIdx.x] = results[threadIdx.y][threadIdx.x * 8 + j];

                    if (vals_iter == 0 && threadIdx.x == 0) {
                        printf("vals_iter:%d vals_to_scatter[%d, %d]: %lf\n", vals_iter, threadIdx.x, threadIdx.y, vals_to_scatter[threadIdx.y][i + j * 4 + threadIdx.x]);
                    }
                }
            }
        }
        __syncwarp();
        if (threadIdx.x < 4) {
            p = 0;
            vals_iter = 0;
            for (int i = 0; i < level - 1; i++)
            {
                int layer_items = (level - i) * (level - i - 1) / 2;
                for (int j = 0; j < level - i - 1; j++)
                {
                    p++;
                    for (int k = 1; k < level - i - j - 1; k++)
                    {
                        int e[4] = {p, p + layer_items - j, p + level - i - j, p + 1};
                        vecY[e[threadIdx.x] * stride + macro_idx] = vals_to_scatter[threadIdx.y][4 * vals_iter + threadIdx.x];
                        vals_iter += 1;
                        p++;
                    }
                    p++;
                }
                p++;
            }
        }

        // Second case
        jacobian_to_laplacian(macro_J, micro_L, tetra_level, 1);
        // wmma::load_matrix_sync(a_frag, micro_L, 4);

        // if (e == 0) {
        //     printf("Laplacian of Category %d\n", 1);
        //     print_matrix(micro_L, 4, 4);
        // }

        p = 0;
        if (threadIdx.x < 4) {
            for (int i = 0; i < level - 1; i++)
            {
                int layer_items = (level - i) * (level - i - 1) / 2;
                for (int j = 0; j < level - i - 1; j++)
                {
                    p++;
                    for (int k = 1; k < level - i - j - 1; k++)
                    {
                        int e[4] = {
                            p, 
                            p + layer_items + level - i - j - 1 + level - i - j - 1,
                            p + layer_items + level - i - j, 
                            p + layer_items + level - i - j - 1
                        };
                        vals_gathered[threadIdx.y][4 * vals_iter + threadIdx.x] = vecX[e[threadIdx.x] * stride + macro_idx];
                        vals_iter += 1;
                        p++;
                    }
                    p++;
                }
                p++;
            }
        }

        __syncwarp();
        for (int i = 0; i < vals_iter; i += 32) {
            // Load the inputs
            // wmma::load_matrix_sync(b_frag, &vals_gathered[threadIdx.y][i], 8);
            // Perform the matrix multiplication
            // wmma::mma_sync(c_frag, a_frag, b_frag, c_frag);
            // Store the output (sync is necessary for &vals_to_scatter[i] due to padding overwritting memory)
            // wmma::store_matrix_sync(results[threadIdx.y], c_frag, 8, // wmma::mem_row_major);
            if (threadIdx.x < 4) {
                for (int j = 0; j < 8; j += 1) {
                    vals_to_scatter[threadIdx.y][i + j * 4 + threadIdx.x] = results[threadIdx.y][threadIdx.x * 8 + j];
                }
            }
        }
        __syncwarp();
        if (threadIdx.x < 4) {
            p = 0;
            vals_iter = 0;

            for (int i = 0; i < level - 1; i++)
            {
                int layer_items = (level - i) * (level - i - 1) / 2;
                for (int j = 0; j < level - i - 1; j++)
                {
                    p++;
                    for (int k = 1; k < level - i - j - 1; k++)
                    {
                        int e[4] = {
                            p, 
                            p + layer_items + level - i - j - 1 + level - i - j - 1,
                            p + layer_items + level - i - j, 
                            p + layer_items + level - i - j - 1
                        };
                        vecY[e[threadIdx.x] * stride + macro_idx] = vals_to_scatter[threadIdx.y][4 * vals_iter + threadIdx.x];
                        vals_iter += 1;
                        p++;
                    }
                    p++;
                }
                p++;
            }
        }

        jacobian_to_laplacian(macro_J, micro_L, tetra_level, 2);
        // wmma::load_matrix_sync(a_frag, micro_L, 4);

        // if (e == 0) {
        //     printf("Laplacian of Category %d\n", 2);
        //     print_matrix(micro_L, 4, 4);
        // }

        // Third case
        p = 0;
        if (threadIdx.x < 4) {
            for (int i = 0; i < level - 1; i++)
            {
                int layer_items = (level - i) * (level - i - 1) / 2;
                for (int j = 0; j < level - i - 1; j++)
                {
                    p++;
                    for (int k = 1; k < level - i - j - 1; k++)
                    {
                        int e[4] = {
                            p, 
                            p + layer_items + level - i - j,
                            p + layer_items + level - i - j - 1 + level - i - j - 1,
                            p + level - i - j
                        };
                        vals_gathered[threadIdx.y][4 * vals_iter + threadIdx.x] = vecX[e[threadIdx.x] * stride + macro_idx];
                        vals_iter += 1;
                        p++;
                    }
                    p++;
                }
                p++;
            }
        }

        __syncwarp();
        for (int i = 0; i < vals_iter; i += 32) {
            // Load the inputs
            // wmma::load_matrix_sync(b_frag, &vals_gathered[threadIdx.y][i], 8);
            // Perform the matrix multiplication
            // wmma::mma_sync(c_frag, a_frag, b_frag, c_frag);
            // Store the output (sync is necessary for &vals_to_scatter[i] due to padding overwritting memory)
            // wmma::store_matrix_sync(results[threadIdx.y], c_frag, 8, // wmma::mem_row_major);
            if (threadIdx.x < 4) {
                for (int j = 0; j < 8; j += 1) {
                    vals_to_scatter[threadIdx.y][i + j * 4 + threadIdx.x] = results[threadIdx.y][threadIdx.x * 8 + j];
                }
            }
        }
        __syncwarp();
        if (threadIdx.x < 4) {
            p = 0;
            vals_iter = 0;

            for (int i = 0; i < level - 1; i++)
            {
                int layer_items = (level - i) * (level - i - 1) / 2;
                for (int j = 0; j < level - i - 1; j++)
                {
                    p++;
                    for (int k = 1; k < level - i - j - 1; k++)
                    {
                        int e[4] = {
                            p, 
                            p + layer_items + level - i - j,
                            p + layer_items + level - i - j - 1 + level - i - j - 1,
                            p + level - i - j
                        };
                        vecY[e[threadIdx.x] * stride + macro_idx] = vals_to_scatter[threadIdx.y][4 * vals_iter + threadIdx.x];
                        vals_iter += 1;

                        p++;
                    }
                    p++;
                }
                p++;
            }
        }

        jacobian_to_laplacian(macro_J, micro_L, tetra_level, 3);
        // wmma::load_matrix_sync(a_frag, micro_L, 4);
        // if (e == 0) {
        //     printf("Laplacian of Category %d\n", 3);
        //     print_matrix(micro_L, 4, 4);
        // }

        // Fourth case
        p = 0;

        if (threadIdx.x < 4) {
            for (int i = 0; i < level - 1; i++)
            {
                int layer_items = (level - i) * (level - i - 1) / 2;
                for (int j = 0; j < level - i - 1; j++)
                {
                    p++;
                    for (int k = 1; k < level - i - j - 1; k++)
                    {
                        int e[4] = {
                            p, 
                            p + layer_items + level - i - j - 1 + level - i - j - 1,
                            p + layer_items + level - i - j - 1,
                            p + level - i - j - 1
                        };
                        vals_gathered[threadIdx.y][4 * vals_iter + threadIdx.x] = vecX[e[threadIdx.x] * stride + macro_idx];
                        vals_iter += 1;
                        p++;
                    }
                    p++;
                }
                p++;
            }
        }

        __syncwarp();
        for (int i = 0; i < vals_iter; i += 32) {
            // Load the inputs
            // wmma::load_matrix_sync(b_frag, &vals_gathered[threadIdx.y][i], 8);
            // Perform the matrix multiplication
            // wmma::mma_sync(c_frag, a_frag, b_frag, c_frag);
            // Store the output (sync is necessary for &vals_to_scatter[i] due to padding overwritting memory)
            // wmma::store_matrix_sync(results[threadIdx.y], c_frag, 8, // wmma::mem_row_major);
            if (threadIdx.x < 4) {
                for (int j = 0; j < 8; j += 1) {
                    vals_to_scatter[threadIdx.y][i + j * 4 + threadIdx.x] = results[threadIdx.y][threadIdx.x * 8 + j];
                }
            }
        }
        __syncwarp();
        if (threadIdx.x < 4) {
            p = 0;
            vals_iter = 0;

            for (int i = 0; i < level - 1; i++)
            {
                int layer_items = (level - i) * (level - i - 1) / 2;
                for (int j = 0; j < level - i - 1; j++)
                {
                    p++;
                    for (int k = 1; k < level - i - j - 1; k++)
                    {
                        int e[4] = {
                            p, 
                            p + layer_items + level - i - j - 1 + level - i - j - 1,
                            p + layer_items + level - i - j - 1,
                            p + level - i - j - 1
                        };
                        vecY[e[threadIdx.x] * stride + macro_idx] = vals_to_scatter[threadIdx.y][4 * vals_iter + threadIdx.x];
                        vals_iter += 1;
                        p++;
                    }
                    p++;
                }
                p++;
            }
        }

        jacobian_to_laplacian(macro_J, micro_L, tetra_level, 4);
        // wmma::load_matrix_sync(a_frag, micro_L, 4);
        // if (e == 0) {
        //     printf("Laplacian of Category %d\n", 4);
        //     print_matrix(micro_L, 4, 4);
        // }

        // Fifth case
        p = 0;
        if (threadIdx.x < 4) {

            for (int i = 1; i < level - 1; i++)
            {
                p = p + level - i + 1;
                int layer_items = (level - i) * (level - i - 1) / 2;
                for (int j = 0; j < level - i - 1; j++)
                {
                    p++;
                    for (int k = 1; k < level - i - j - 1; k++)
                    {
                        int e[4] = {
                            p, 
                            p + layer_items + level - i - j + level - i,
                            p + layer_items + level - i,
                            p + layer_items + level - i - j + level - i - 1
                        };
                        vals_gathered[threadIdx.y][4 * vals_iter + threadIdx.x] = vecX[e[threadIdx.x] * stride + macro_idx];
                        vals_iter += 1;
                        p++;
                    }
                    p++;
                }
                p++;
            }
        }

        __syncwarp();
        for (int i = 0; i < vals_iter; i += 32) {
            // Load the inputs
            // wmma::load_matrix_sync(b_frag, &vals_gathered[threadIdx.y][i], 8);
            // Perform the matrix multiplication
            // wmma::mma_sync(c_frag, a_frag, b_frag, c_frag);
            // Store the output (sync is necessary for &vals_to_scatter[i] due to padding overwritting memory)
            // wmma::store_matrix_sync(results[threadIdx.y], c_frag, 8, // wmma::mem_row_major);
            if (threadIdx.x < 4) {
                for (int j = 0; j < 8; j += 1) {
                    vals_to_scatter[threadIdx.y][i + j * 4 + threadIdx.x] = results[threadIdx.y][threadIdx.x * 8 + j];
                }
            }
        }
        __syncwarp();
        if (threadIdx.x < 4) {
            p = 0;
            vals_iter = 0;

            for (int i = 1; i < level - 1; i++)
            {
                p = p + level - i + 1;
                int layer_items = (level - i) * (level - i - 1) / 2;
                for (int j = 0; j < level - i - 1; j++)
                {
                    p++;
                    for (int k = 1; k < level - i - j - 1; k++)
                    {
                        int e[4] = {
                            p, 
                            p + layer_items + level - i - j + level - i,
                            p + layer_items + level - i,
                            p + layer_items + level - i - j + level - i - 1
                        };
                        vecY[e[threadIdx.x] * stride + macro_idx] = vals_to_scatter[threadIdx.y][4 * vals_iter + threadIdx.x];
                        vals_iter += 1;
                        p++;
                    }
                    p++;
                }
                p++;
            }
        }

        jacobian_to_laplacian(macro_J, micro_L, tetra_level, 5);
        // wmma::load_matrix_sync(a_frag, micro_L, 4);
        // if (e == 0) {
        //     printf("Laplacian of Category %d\n", 5);
        //     print_matrix(micro_L, 4, 4);
        // }

        // Sixth case
        p = 0;
        if (threadIdx.x < 4) {

            for (int i = 0; i < level - 1; i++)
            {
                int layer_items = (level - i) * (level - i - 1) / 2;
                for (int j = 0; j < level - i - 1; j++)
                {
                    p++;
                    for (int k = 1; k < level - i - j - 1; k++)
                    {
                        int e[4] = {
                            p, 
                            p + level - i - j,
                            p + layer_items + level - i - j - 1 + level - i - j - 1,
                            p + level - i - j - 1
                        };
                        vals_gathered[threadIdx.y][4 * vals_iter + threadIdx.x] = vecX[e[threadIdx.x] * stride + macro_idx];
                        vals_iter += 1;
                        p++;
                    }
                    p++;
                }
                p++;
            }
        }

        __syncwarp();
        for (int i = 0; i < vals_iter; i += 32) {
            // Load the inputs
            // wmma::load_matrix_sync(b_frag, &vals_gathered[threadIdx.y][i], 8);
            // Perform the matrix multiplication
            // wmma::mma_sync(c_frag, a_frag, b_frag, c_frag);
            // Store the output (sync is necessary for &vals_to_scatter[i] due to padding overwritting memory)
            // wmma::store_matrix_sync(results[threadIdx.y], c_frag, 8, // wmma::mem_row_major);
            if (threadIdx.x < 4) {
                for (int j = 0; j < 8; j += 1) {
                    vals_to_scatter[threadIdx.y][i + j * 4 + threadIdx.x] = results[threadIdx.y][threadIdx.x * 8 + j];
                }
            }
        }
        __syncwarp();
        if (threadIdx.x < 4) {
            p = 0;
            vals_iter = 0;

            for (int i = 0; i < level - 1; i++)
            {
                int layer_items = (level - i) * (level - i - 1) / 2;
                for (int j = 0; j < level - i - 1; j++)
                {
                    p++;
                    for (int k = 1; k < level - i - j - 1; k++)
                    {
                        int e[4] = {
                            p, 
                            p + level - i - j,
                            p + layer_items + level - i - j - 1 + level - i - j - 1,
                            p + level - i - j - 1
                        };
                        vecY[e[threadIdx.x] * stride + macro_idx] = vals_to_scatter[threadIdx.y][4 * vals_iter + threadIdx.x];
                        vals_iter += 1;
                        p++;
                    }
                    p++;
                }
                p++;
            }

        }
    }
}

int compute_nodes_number_aux(int tetra_level)
{
    int num_nodes = 0;
    if (tetra_level % 2 == 0)
    {
        for (int i = 0; i < floor(tetra_level / 2); i++)
        {
            num_nodes += (tetra_level - i + 1) * (i + 1) * 2;
        }
        num_nodes += (tetra_level / 2 + 1) * (tetra_level / 2 + 1);
    }
    else 
    {
        for (int i = 0; i < floor(tetra_level / 2) + 1; i++)
        {
            num_nodes += (tetra_level - i + 1) * (i + 1) * 2;
        }
    }
    return num_nodes;
}

int compute_nodes_number(int tetra_level)
{
    // 1 layer = 4
    // 2 layer = 10
    return (tetra_level + 3) * (tetra_level + 2) * (tetra_level + 1) / 6;
}

int compute_tets_number(int tetra_level)
{
    return (int) pow(tetra_level, 3);
}

// Kernel to apply Dirichlet boundary conditions
__global__ void applyDirichlet(real_t *Ax, real_t *rhs, size_t num_macro_tets, size_t stride, size_t *dirichlet_nodes, size_t num_dirichlet_nodes) {
    for (size_t idx = blockIdx.x * blockDim.x + threadIdx.x; idx < num_macro_tets;
         idx += blockDim.x * gridDim.x) {
            for (int j = 0; j < num_dirichlet_nodes; j += 1) {
                size_t dirichlet_node_idx = dirichlet_nodes[j];
                Ax[dirichlet_node_idx * stride + idx] = rhs[dirichlet_node_idx * stride + idx];
            }
    }
}

// Kernel to compute the residual r = rhs - Ax
__global__ void computeResidual(real_t *r, real_t *rhs, real_t *Ax, size_t num_macro_tets, size_t stride, size_t num_local_nodes) {
    // iterate over some tetrahedrons
    for (size_t macro_tet_idx = blockIdx.x * blockDim.x + threadIdx.x; macro_tet_idx < num_macro_tets;
         macro_tet_idx += blockDim.x * gridDim.x) {
            // iterate over the local nodes
            for (size_t node_idx = 0; node_idx < num_local_nodes; node_idx += 1) {
                r[node_idx * stride + macro_tet_idx] = rhs[node_idx * stride + macro_tet_idx] - Ax[node_idx * stride + macro_tet_idx];
            }
    }
}

// Kernel for vector dot product: result = sum(a[i] * b[i])
__global__ void dotProduct(const real_t* a, const real_t* b, real_t* result, size_t num_macro_tets, size_t stride, size_t num_local_nodes) {
    for (size_t macro_tet_idx = blockIdx.x * blockDim.x + threadIdx.x; macro_tet_idx < num_macro_tets;
         macro_tet_idx += blockDim.x * gridDim.x) {
            // iterate over the local nodes
            for (size_t node_idx = 0; node_idx < num_local_nodes; node_idx += 1) {
                result[macro_tet_idx] += a[node_idx * stride + macro_tet_idx] * b[node_idx * stride + macro_tet_idx];
            }
            if (macro_tet_idx == 0) {
                printf("dotProduct of %d: %lf\n", macro_tet_idx, result[macro_tet_idx]);
            }
    }
}

// Kernel for vector update: y = alpha * x + b
__global__ void vectorAdd(real_t *y, const real_t *alpha, const real_t *x, const real_t *b, size_t stride, size_t num_macro_tets, size_t num_local_nodes) {
    // iterate over some tetrahedrons
    for (size_t macro_tet_idx = blockIdx.x * blockDim.x + threadIdx.x; macro_tet_idx < num_macro_tets;
         macro_tet_idx += blockDim.x * gridDim.x) {
            // iterate over the local nodes
            for (size_t node_idx = 0; node_idx < num_local_nodes; node_idx += 1) {
                y[node_idx * stride + macro_tet_idx] = alpha[macro_tet_idx] * x[node_idx * stride + macro_tet_idx] + b[node_idx * stride + macro_tet_idx];
            }

            if (macro_tet_idx == 0) {
                printf("vecX after vectorAdd: \n");
                for (int n = 0; n < 100; n += 1) {
                    printf("%lf ", y[n * stride + macro_tet_idx]);
                }
                printf("\n");
            }
    }

}

// Kernel for vector update: x += alpha * r 
__global__ void vectorUpdate(real_t *x, const real_t alpha, const real_t *r, size_t stride, size_t num_macro_tets, size_t num_local_nodes) {
    // iterate over some tetrahedrons
    for (size_t macro_tet_idx = blockIdx.x * blockDim.x + threadIdx.x; macro_tet_idx < num_macro_tets;
         macro_tet_idx += blockDim.x * gridDim.x) {
            // iterate over the local nodes
            for (size_t node_idx = 0; node_idx < num_local_nodes; node_idx += 1) {
                x[node_idx * stride + macro_tet_idx] = alpha * r[node_idx * stride + macro_tet_idx];
            }
    }

}

// Kernel for vector update: x = x - alpha * p
__global__ void vectorMinus(real_t* x, const real_t* p, real_t *alpha, size_t stride, size_t num_macro_tets, size_t num_local_nodes) {
    // iterate over some tetrahedrons
    for (size_t macro_tet_idx = blockIdx.x * blockDim.x + threadIdx.x; macro_tet_idx < num_macro_tets;
         macro_tet_idx += blockDim.x * gridDim.x) {
            // iterate over the local nodes
            for (size_t node_idx = 0; node_idx < num_local_nodes; node_idx += 1) {
                x[node_idx * stride + macro_tet_idx] -= alpha[macro_tet_idx] * p[node_idx * stride + macro_tet_idx];
            }

            if (macro_tet_idx == 0) {
                printf("p in vectorMinus: \n");
                for (int n = 0; n < 100; n += 1) {
                    printf("%lf ", p[n * stride + macro_tet_idx]);
                }
                printf("\n");
                printf("alpha: %lf\n", alpha[macro_tet_idx]);
            }
    }

}

// Kernel for division update: alpha = up / down
__global__ void scalarDivide(real_t* alpha, const real_t* up, real_t *down, size_t num_macro_tets) {
    // iterate over some tetrahedrons
    for (size_t macro_tet_idx = blockIdx.x * blockDim.x + threadIdx.x; macro_tet_idx < num_macro_tets;
         macro_tet_idx += blockDim.x * gridDim.x) {
            // iterate over the local nodes
            alpha[macro_tet_idx] = up[macro_tet_idx] / down[macro_tet_idx];
            if (macro_tet_idx == 0) {
                printf("scalarDivide of %lf/%lf: %lf\n", up[macro_tet_idx], down[macro_tet_idx], alpha[macro_tet_idx]);
            }
    }

}

__global__ void checkConvergence(const real_t tol, const real_t* residual, int num_macro_tets, size_t* converged) {
    for (size_t macro_tet_idx = blockIdx.x * blockDim.x + threadIdx.x; macro_tet_idx < num_macro_tets;
         macro_tet_idx += blockDim.x * gridDim.x) {
        if (residual[macro_tet_idx] >= tol * tol) {
            *converged = 0;
            return;
        }
    }
}

// CUDA Kernel to set the Dirichlet boundary conditions
__global__ void setDirichletBoundaryConditions(size_t *dirichlet_nodes, real_t *rhs, real_t *x, size_t num_macro_tets, size_t stride, real_t *dirichlet_values, size_t num_dirichlet_nodes) {
    for (size_t macro_tet_idx = blockIdx.x * blockDim.x + threadIdx.x; macro_tet_idx < num_macro_tets;
         macro_tet_idx += blockDim.x * gridDim.x) {
            // iterate over the local nodes
            for (size_t i = 0; i < num_dirichlet_nodes; i += 1) {
                size_t local_node_idx = dirichlet_nodes[i];
                rhs[local_node_idx * stride + macro_tet_idx] = dirichlet_values[i];
                x[local_node_idx * stride + macro_tet_idx] = dirichlet_values[i];
            }
    }
}

void set_boundary_conditions_cuda(size_t num_nodes, real_t *rhs, real_t *x, size_t num_macro_tets, size_t stride, size_t **dirichlet_nodes, size_t *num_dirichlet_nodes)
{
    *num_dirichlet_nodes = 2;
    checkCudaError(hipMalloc(dirichlet_nodes, (*num_dirichlet_nodes) * sizeof(size_t)));

    // Set the Dirichlet nodes (e.g., first and last nodes)
    size_t h_dirichlet_nodes[] = {0, num_nodes - 1};
    checkCudaError(hipMemcpy(*dirichlet_nodes, h_dirichlet_nodes, (*num_dirichlet_nodes) * sizeof(size_t), hipMemcpyHostToDevice));

    // Set the Dirichlet values corresponding to the Dirichlet nodes
    real_t h_dirichlet_values[] = {1.0, 0.0};

    real_t *d_dirichlet_values;
    checkCudaError(hipMalloc(&d_dirichlet_values, (*num_dirichlet_nodes) * sizeof(real_t)));
    checkCudaError(hipMemcpy(d_dirichlet_values, h_dirichlet_values, (*num_dirichlet_nodes) * sizeof(real_t), hipMemcpyHostToDevice));

    // Launch the kernel to set the Dirichlet boundary conditions
    int threadsPerBlock = BLOCK_SIZE;
    int numBlocks = (num_macro_tets + threadsPerBlock - 1) / threadsPerBlock;
    setDirichletBoundaryConditions<<<numBlocks, threadsPerBlock>>>(*dirichlet_nodes, rhs, x, num_macro_tets, stride, d_dirichlet_values, *num_dirichlet_nodes);

    ifLastErrorExists("Kernel launch failed");

    // Free the temporary device memory for Dirichlet values
    checkCudaError(hipFree(d_dirichlet_values));
}

__host__ real_t *solve_using_gradient_descent(int tetra_level, int num_macro_tets, int num_nodes, real_t *macro_jacobians)
{
    // Allocate variables for boundary conditions
    int max_iter = 100;
    double tol = 1e-2;
    real_t *h_x, *h_r;
    checkCudaError(hipHostMalloc(&h_x, num_macro_tets * sizeof(real_t) * num_nodes));
    checkCudaError(hipHostMalloc(&h_r, num_macro_tets * sizeof(real_t) * num_nodes));

    // Allocate GPU memory
    real_t *d_b, *d_x, *d_r, *d_Ax;
    checkCudaError(hipMalloc(&d_b, num_macro_tets * num_nodes * sizeof(real_t)));
    checkCudaError(hipMalloc(&d_x, num_macro_tets * num_nodes * sizeof(real_t)));
    checkCudaError(hipMalloc(&d_Ax, num_macro_tets * num_nodes * sizeof(real_t)));
    checkCudaError(hipMalloc(&d_r, num_macro_tets * num_nodes * sizeof(real_t)));

    dim3 tensorCoreBlockDim(32, BLOCK_Y_SIZE, 1);
    int numTensorCoreBlocks = (num_macro_tets + tensorCoreBlockDim.y - 1) / tensorCoreBlockDim.y;

    hipblasHandle_t cublas_handle;
    hipblasCreate(&cublas_handle);

    size_t *d_dirichlet_nodes;
    size_t num_dirichlet_nodes;

    int stride = num_macro_tets;

    set_boundary_conditions_cuda(num_nodes, d_b, d_x, num_macro_tets, stride, &d_dirichlet_nodes, &num_dirichlet_nodes);
    checkCudaError(hipMemcpy(h_x, d_x, sizeof(real_t *) * num_macro_tets * num_nodes, hipMemcpyDeviceToHost));

    int threadsPerBlock = BLOCK_SIZE;
    int numBlocks = (num_macro_tets + threadsPerBlock - 1) / threadsPerBlock;

    // Start Gradient Descent iterations
    int iter = 0;
    real_t gamma = 7 * 1e-1;
    while (iter < max_iter) {

        // Initialize r = b - A * x
        cu_macro_tet4_laplacian_apply_kernel<<<numTensorCoreBlocks, tensorCoreBlockDim>>>(num_macro_tets, stride, tetra_level, macro_jacobians, d_x, d_Ax);
        ifLastErrorExists("Kernel launch failed");

        applyDirichlet<<<numBlocks, threadsPerBlock>>>(d_Ax, d_b, num_macro_tets, stride, d_dirichlet_nodes, num_dirichlet_nodes);
        ifLastErrorExists("Kernel launch failed");

        computeResidual<<<numBlocks, threadsPerBlock>>>(d_r, d_b, d_Ax, num_macro_tets, stride, num_nodes);
        ifLastErrorExists("Kernel launch failed");

        printf("Iteration: %d, computeResidual finished\n", iter);
        hipDeviceSynchronize();

        // cuBLAS for reduction
        // minSquareError computeNorm
        double norm_r = 0;
        hipblasDnrm2(cublas_handle, num_macro_tets * num_nodes, d_r, 1, &norm_r);
        ifLastErrorExists("Kernel launch failed");

        printf("Iteration: %d, Global 2-norm = %lf\n", iter, norm_r);

        // Check for convergence
        if (norm_r < tol) {
            checkCudaError(hipMemcpy(&h_x, d_x, sizeof(real_t) * num_nodes * num_macro_tets, hipMemcpyDeviceToHost));
            for (int n = 0; n < num_nodes * num_macro_tets; n += num_macro_tets) {
                printf("%lf ", h_x[n]);
            }
            printf("Converged after %d iterations, 2-norm: %lf\n", iter, norm_r);
            // hipFree(converged);
            break;
        }

        // Update x = x + alpha * p
        vectorUpdate<<<numBlocks, threadsPerBlock>>>(d_x, gamma, d_r, stride, num_macro_tets, num_nodes);
        ifLastErrorExists("Kernel launch failed");
        
        // checkCudaError(hipMemcpy(h_x, d_x, sizeof(real_t *) * num_macro_tets * num_nodes, hipMemcpyDeviceToHost));
        // printf("resulting x from vectorAdd: \n");
        // for (int n = 0; n < num_nodes * num_macro_tets; n += num_macro_tets) {
        //     printf("%lf ", h_x[n]);
        // }
        // printf("\n");

        iter++;
    }

    // Free GPU memory
    checkCudaError(hipFree(d_b));
    checkCudaError(hipFree(d_x));
    checkCudaError(hipFree(d_r));
    checkCudaError(hipFree(d_Ax));

    // Free allocated memory
    checkCudaError(hipFree(d_dirichlet_nodes));

    checkCudaError(hipHostFree(h_r));

    hipblasDestroy(cublas_handle);

    return h_x;
}

void compute_A(real_t *p0, real_t *p1, real_t *p2, real_t *p3, real_t *A)
{
    for (int i = 0; i < 3; i++)
    {
        A[i] = p1[i] - p0[i];
        A[3 + i] = p2[i] - p0[i];
        A[6 + i] = p3[i] - p0[i];
    }
    // assert(determinant_3x3(A) > 0);
}

int main(void) {
    int tetra_level = 8;

    // Compute the number of nodes
    int num_nodes = compute_nodes_number(tetra_level);
    int num_micro_tets = compute_tets_number(tetra_level);

    int num_macro_tets = 1000;

    real_t *macro_jacobians, *h_macro_jacobians;
    checkCudaError(hipHostMalloc(&h_macro_jacobians, sizeof(real_t) * 9 * num_macro_tets));
    checkCudaError(hipMalloc(&macro_jacobians, sizeof(real_t) * 9 * num_macro_tets));

    for (int i = 0; i < num_macro_tets; i += 1) {
        real_t macro_J[9];
        real_t p0[3] = {0, 0, 0};
        real_t p1[3] = {1, 0, 0};
        real_t p2[3] = {0, 1, 0};
        real_t p3[3] = {0, 0, 1};
        compute_A(p0, p1, p2, p3, macro_J);
        for (int j = 0; j < 9; j += 1) {
            h_macro_jacobians[j * num_macro_tets + i] = macro_J[j];
        }
    }

    checkCudaError(hipMemcpy(macro_jacobians, h_macro_jacobians, 9 * sizeof(real_t) * num_macro_tets, hipMemcpyHostToDevice));
    real_t *h_x = solve_using_gradient_descent(tetra_level, num_macro_tets, num_nodes, macro_jacobians);
    checkCudaError(hipHostFree(h_x));

    checkCudaError(hipHostFree(h_macro_jacobians));
    checkCudaError(hipFree(macro_jacobians));
    // solve_using_gradient_descent(tetra_level, num_nodes, num_micro_tets, macro_J);

    return 0;

}