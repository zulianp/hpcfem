
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h> // cudaMalloc, cudaMemcpy, etc.
#include <hipsparse.h>         // cusparseSpMV
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE

#define CHECK_CUDA(func)                                                       \
{                                                                              \
    hipError_t status = (func);                                               \
    if (status != hipSuccess) {                                               \
        printf("CUDA API failed at line %d with error: %s (%d)\n",             \
               __LINE__, hipGetErrorString(status), status);                  \
        return EXIT_FAILURE;                                                   \
    }                                                                          \
}

#define CHECK_CUSPARSE(func)                                                   \
{                                                                              \
    hipsparseStatus_t status = (func);                                          \
    if (status != HIPSPARSE_STATUS_SUCCESS) {                                   \
        printf("CUSPARSE API failed at line %d with error: %s (%d)\n",         \
               __LINE__, hipsparseGetErrorString(status), status);              \
        return EXIT_FAILURE;                                                   \
    }                                                                          \
}

int* load_int32_array(const char *filename, int64_t *elements_read) {
    FILE *file;
    long file_size;
    int *buffer;
    size_t num_elements;

    // Open the file for reading in binary mode
    file = fopen(filename, "rb");
    if (file == NULL) {
        perror("Error opening file");
        return NULL;
    }

    // Seek to the end of the file to get its size
    fseek(file, 0, SEEK_END);
    file_size = ftell(file);
    rewind(file);

    // Calculate the number of elements based on the file size
    num_elements = file_size / sizeof(int);

    // Allocate managed memory for the buffer
    hipError_t err = hipMallocManaged((void**)&buffer, file_size);
    if (err != hipSuccess) {
        fprintf(stderr, "Error allocating managed memory: %s\n", hipGetErrorString(err));
        fclose(file);
        return NULL;
    }

    // Read the entire content of the file into the buffer
    *elements_read = fread(buffer, sizeof(int), num_elements, file);
    if (*elements_read != num_elements) {
        perror("Error reading file");
        hipFree(buffer);
        fclose(file);
        return NULL;
    }

    // Close the file
    fclose(file);

    // Report the number of elements read
    printf("Number of elements read from %s: %d\n", filename, *elements_read);

    return buffer;
}

double* load_float64_array(const char *filename, int64_t *elements_read) {
    FILE *file;
    long file_size;
    double *buffer;
    size_t num_elements;

    // Open the file for reading in binary mode
    file = fopen(filename, "rb");
    if (file == NULL) {
        perror("Error opening file");
        return NULL;
    }

    // Seek to the end of the file to get its size
    fseek(file, 0, SEEK_END);
    file_size = ftell(file);
    rewind(file);

    // Calculate the number of elements based on the file size
    num_elements = file_size / sizeof(double);

    // Allocate managed memory for the buffer
    hipError_t err = hipMallocManaged((void**)&buffer, file_size);
    if (err != hipSuccess) {
        fprintf(stderr, "Error allocating managed memory: %s\n", hipGetErrorString(err));
        fclose(file);
        return NULL;
    }

    // Read the entire content of the file into the buffer
    *elements_read = fread(buffer, sizeof(double), num_elements, file);
    if (*elements_read != num_elements) {
        perror("Error reading file");
        hipFree(buffer);
        fclose(file);
        return NULL;
    }

    // Close the file
    fclose(file);

    // Report the number of elements read
    printf("Number of elements read from %s: %d\n", filename, *elements_read);

    return buffer;
}

int main(void) {
    // Host problem definition
    int A_num_rows      = 1;
    int A_num_cols      = 1;
    int A_nnz           = 1;
    float     alpha           = 1.0f;
    float     beta            = 0.0f;
    //--------------------------------------------------------------------------
    // Device memory management
    // int   *dA_csrOffsets, *dA_columns;
    double *dA_values, *dX, *dY;

    int64_t sellValuesSize = 0;
    int64_t elements_read;

    int *sellSliceOffsets = load_int32_array("sell_slice_offsets.i32", &elements_read);
    double *sellValues = load_float64_array("sell_values.f64", &sellValuesSize);
    int *sellColInd = load_int32_array("sell_column_indices.i32", &elements_read);
    int *sellMetaInfo = load_int32_array("sell_meta.i32", &elements_read);

    int sliceSize = 2;

    A_num_rows = sellMetaInfo[0];
    A_num_cols = sellMetaInfo[0];
    A_nnz = sellMetaInfo[1];
    // sliceSize = sellMetaInfo[2];

    // Timing variables
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    //--------------------------------------------------------------------------
    // CUSPARSE APIs
    hipsparseHandle_t     handle = NULL;
    hipsparseSpMatDescr_t matA;
    hipsparseDnVecDescr_t vecX, vecY;
    void*                dBuffer    = NULL;
    size_t               bufferSize = 0;
    CHECK_CUSPARSE( hipsparseCreate(&handle) )
    // Create sparse matrix A in SELL format

    CHECK_CUSPARSE( cusparseCreateSlicedEll(&matA, A_num_rows, A_num_cols, A_nnz,
                            sellValuesSize, sliceSize, sellSliceOffsets, sellColInd, sellValues,
                            HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                            HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F) )

    CHECK_CUDA( hipMalloc((void**) &dX, A_num_cols * sizeof(double)) )
    CHECK_CUDA( hipMalloc((void**) &dY, A_num_rows * sizeof(double)) )

    // Create dense vector X
    CHECK_CUSPARSE( hipsparseCreateDnVec(&vecX, A_num_cols, dX, HIP_R_64F) )
    // Create dense vector y
    CHECK_CUSPARSE( hipsparseCreateDnVec(&vecY, A_num_rows, dY, HIP_R_64F) )
    // allocate an external buffer if needed
    CHECK_CUSPARSE( hipsparseSpMV_bufferSize(
                                 handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, matA, vecX, &beta, vecY, HIP_R_64F,
                                 HIPSPARSE_SPMV_ALG_DEFAULT, &bufferSize) )
    CHECK_CUDA( hipMalloc(&dBuffer, bufferSize) )

    hipEventRecord(start);

    // execute SpMV
    CHECK_CUSPARSE( hipsparseSpMV(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, matA, vecX, &beta, vecY, HIP_R_64F,
                                 HIPSPARSE_SPMV_ALG_DEFAULT, dBuffer) )

    hipEventRecord(stop);

    // Wait for the event to complete
    hipEventSynchronize(stop);

    // Calculate elapsed time
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    
    double avg_time = milliseconds/1000;
    double avg_throughput = (A_num_rows / avg_time) * 1e-6;

    printf("Time for matrix-vector multiplication: %f milliseconds\n", milliseconds);
    printf("Throughput %g (MDOF/s)\n");

    // destroy matrix/vector descriptors
    CHECK_CUSPARSE( hipsparseDestroySpMat(matA) )
    CHECK_CUSPARSE( hipsparseDestroyDnVec(vecX) )
    CHECK_CUSPARSE( hipsparseDestroyDnVec(vecY) )
    CHECK_CUSPARSE( hipsparseDestroy(handle) )
    //--------------------------------------------------------------------------
    // device result check

    // CHECK_CUDA( cudaMemcpy(hY, dY, A_num_rows * sizeof(float),
    //                        cudaMemcpyDeviceToHost) )
    // int correct = 1;
    // for (int i = 0; i < A_num_rows; i++) {
    //     if (hY[i] != hY_result[i]) { // direct floating point comparison is not
    //         correct = 0;             // reliable
    //         break;
    //     }
    // }
    // if (correct)
    //     printf("spmv_csr_example test PASSED\n");
    // else
    //     printf("spmv_csr_example test FAILED: wrong result\n");

    //--------------------------------------------------------------------------
    // device memory deallocation
    // CHECK_CUDA( cudaFree(dBuffer) )
    // CHECK_CUDA( cudaFree(dA_csrOffsets) )
    // CHECK_CUDA( cudaFree(dA_columns) )
    // CHECK_CUDA( cudaFree(dA_values) )
    CHECK_CUDA( hipFree(dX) )
    CHECK_CUDA( hipFree(dY) )
    return EXIT_SUCCESS;
}

